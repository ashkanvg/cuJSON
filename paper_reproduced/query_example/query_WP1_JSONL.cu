#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <thread>
#include <x86intrin.h>
#include <string.h>
#include <bitset>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/gather.h>
#include <thrust/extrema.h>
#include <thrust/partition.h>
#include <thrust/execution_policy.h>
#include <inttypes.h>
#include "../src/query/query_iterator.cpp"
#include <thrust/host_vector.h>
#include <>
#include <vector>
#include <cstring>



// #include "./12-GJSON-Class.cuh"

#define         MAXLINELENGTH     268435456   //4194304 8388608 33554432 67108864 134217728 201326592 268435456 536870912 805306368 1073741824// Max record size
                                              //4MB       8MB     32BM    64MB      128MB    192MB     256MB     512MB     768MB       1GB
#define         BUFSIZE           268435456   //4194304 8388608 33554432 67108864 134217728 201326592 268435456 536870912 805306368 1073741824

#define BLOCKSIZE 256

#define OPENBRACKET 91
#define CLOSEBRACKET 93
#define OPENBRACE 123
#define CLOSEBRACE 125
#define I 73

#define ROW1 1
#define ROW2 2
#define ROW3 3
#define ROW4 4
#define ROW5 5

using namespace std;
using namespace std::chrono;


struct inputStartStruct{
    uint32_t  size;
    int       result_size;
    uint8_t*  block;
    int32_t*  res;
    int lastChunkIndex;
    int lastStructuralIndex;
};

struct time_cost_EE{
    float EE_t;
    float EE_t_val;
    float EE_t_tok;
    float EE_t_pars;
    float copy_start;
    float copy_start_total;
    float copy_end;
    float copy_end_toal;
    float EE_total; 
};
struct time_cost_EE time_EE = {0,0,0,0,0,0,0,0,0};


struct is_one{
    __host__ __device__
    bool operator()(const int x){return (x == 1);}
};
struct is_opening{
  __host__ __device__ 
  bool operator()(char x){return (x==OPENBRACE) || (x==OPENBRACKET);}
};
struct is_closing{
  __host__ __device__
  bool operator()(char x){return (x==CLOSEBRACE) || (x==CLOSEBRACKET);}
};
struct decrease{
  __host__ __device__ 
  int operator()(int x){return x-1;}
};
struct increase{
  __host__ __device__ 
  int operator()(int x){return x++;}
};

__device__ __forceinline__
uint32_t prefix_xor(uint32_t x) {
    x ^= (x << 1);
    x ^= (x << 2);
    x ^= (x << 4);
    x ^= (x << 8);
    x ^= (x << 16);
    return x;
}
__device__ __forceinline__
uint64_t prefix_xor64(uint64_t x) {
    x ^= (x << 1);
    x ^= (x << 2);
    x ^= (x << 4);
    x ^= (x << 8);
    x ^= (x << 16);
    x ^= (x << 32);
    return x;
}

__device__
const char* byteToBinary(uint8_t byte) {
    __shared__ char binary[9]; // Shared among threads in the same block, make sure this does not cause race conditions!
    binary[8] = '\0'; // Null terminator for the string

    for (int i = 7; i >= 0; --i) {
        binary[i] = (byte & 0x01) ? '1' : '0';
        byte >>= 1; // Shift right by one bit to process the next bit in the next iteration
    }

    return binary;
}
__device__
void u32ToBinary(uint32_t num, char* out) {
    out[32] = '\0'; // Null-terminator for the string
    for (int i = 31; i >= 0; --i) {
        out[i] = (num & 0x01) ? '1' : '0';
        num >>= 1; // Shift right to get the next bit
    }
}

void print_d32(uint32_t* d_data, int total_padded_32, int rows) {
    uint32_t* h_data = (uint32_t*)malloc(total_padded_32 * rows * sizeof(uint32_t));
    if (!h_data) {
        std::cerr << "Failed to allocate host memory!" << std::endl;
        return;
    }

    hipMemcpy(h_data, d_data, total_padded_32 * rows * sizeof(uint32_t), hipMemcpyDeviceToHost);

    for (int i = 0; i < total_padded_32 * rows; ++i) {
        uint32_t value = h_data[i];
        for (int j = 0; j < 32; ++j) {
            std::cout << ((value >> j) & 1);
        }
        std::cout << std::endl;
    }

    free(h_data);
}

int print_d(uint32_t* input_GPU, int length, int rows){
    uint32_t * input;
    input = (uint32_t*) malloc(sizeof(uint32_t)*length*rows);
    hipMemcpyAsync(input, input_GPU, sizeof(uint32_t)*length*rows, hipMemcpyDeviceToHost);
    
    for(long i =0; i<rows; i++){
      for(long j=401; j<470 && j<length; j++){
        std::bitset<32> y(*(input+j+(i*length)));
        if(j == 129) printf("----129----");
        std::cout << y << ' ';
      }
      cout << "\n";
    }
    free(input);
    return 1;
}
int print8(uint8_t* input, int length, int rows){
    for(long i =0; i<rows; i++){
        for(long j=0; j<length && j<200; j++){
            std::cout << *(input+j+(i*length)) << ' ';
        }
        std::cout << std::endl;
    }
    return 1;
}
int print32(int32_t* input, int length, int rows){
    for(long i =0; i<rows; i++){
        for(long j=0; j<length && j<200; j++){
            std::cout << *(input+j+(i*length)) << ' ';
        }
        std::cout << std::endl;
    }
    return 1;
}
template<typename T>
int print8_d(uint8_t* input_GPU, int length, int rows){

    uint8_t * input;
    input = (uint8_t*) malloc(sizeof(uint8_t)*length);
    hipMemcpyAsync(input, input_GPU, sizeof(uint8_t)*length, hipMemcpyDeviceToHost);

    for(long i =0; i<rows; i++){
        for(long j=0; j<300 && j<length; j++){
            std::cout << (T )*(input+j+(i*length)) << ' ';
        }
        std::cout << std::endl;
    }
    free(input);
    return 1;
}
// Function to print a char array from the GPU
void printCharArrayFromGPU(const char* input_GPU, size_t size) {
    // Allocate memory on the host
    char* input_CPU = (char*)malloc(size + 1);  // +1 for null-terminator

    // Copy data from the device to the host
    hipMemcpy(input_CPU, input_GPU, size, hipMemcpyDeviceToHost);

    // Add null-terminator to make it a valid C-string
    input_CPU[size] = '\0';

    // Print the string
    std::cout << input_CPU << std::endl;

    // Free the host memory
    free(input_CPU);
}
// Function to print int32_t array from the GPU
void printInt8ArrayFromGPU(int8_t* input_GPU, int size) {
    // Allocate memory on the host
    int8_t* input_CPU = (int8_t*) malloc(size * sizeof(int8_t));

    // Copy data from the device to the host
    hipMemcpy(input_CPU, input_GPU, size * sizeof(int8_t), hipMemcpyDeviceToHost);

    // Print the array
    for (int i = 0; i < size; ++i) {
        // std::cout << input_CPU[i] << " ";
        printf("%d ",input_CPU[i]);
    }
    std::cout << std::endl;

    // Free the host memory
    free(input_CPU);
}

// Function to print int32_t array from the GPU
void printUInt8ArrayFromGPU(uint8_t* input_GPU, int size) {
    // Allocate memory on the host
    uint8_t* input_CPU = (uint8_t*) malloc(size * sizeof(uint8_t));

    // Copy data from the device to the host
    hipMemcpy(input_CPU, input_GPU, size * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Print the array
    for (int i = 0; i < size; ++i) {
        // std::cout << input_CPU[i] << " ";
        printf("%d ",input_CPU[i]);
    }
    std::cout << std::endl;

    // Free the host memory
    free(input_CPU);
}

// Function to print int32_t array from the GPU
void printInt32ArrayFromGPU(const int32_t* input_GPU, size_t size) {
    // Allocate memory on the host
    int32_t* input_CPU = (int32_t*)malloc(size * sizeof(int32_t));

    // Copy data from the device to the host
    hipMemcpy(input_CPU, input_GPU, size * sizeof(int32_t), hipMemcpyDeviceToHost);

    // Print the array
    for (size_t i = 0; i < size; ++i) {
        std::cout << input_CPU[i] << " ";
    }
    std::cout << std::endl;

    // Free the host memory
    free(input_CPU);
}

// Function to print int32_t array from the GPU
void printUInt32ArrayFromGPU(const uint32_t* input_GPU, size_t size) {
    // Allocate memory on the host
    uint32_t* input_CPU = (uint32_t*)malloc(size * sizeof(uint32_t));

    // Copy data from the device to the host
    hipMemcpy(input_CPU, input_GPU, size * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Print the array
    for (size_t i = 0; i < size; ++i) {
        std::cout << input_CPU[i] << " ";
    }
    std::cout << std::endl;

    // Free the host memory
    free(input_CPU);
}


void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        exit(1);
    }
}


// prev1            --> 4 character
// result           --> source
// size             --> total size of array
// total_padded_32  --> based on size howmany thread work on that
__device__ __forceinline__
void vectorizedClassification(uint32_t block_compressed, uint32_t prev1, uint32_t& result, uint64_t size, int total_padded_32){
    constexpr const uint8_t TOO_SHORT   = 1<<0; // 00000001
                                                // The leading byte must be followed by N-1 continuation bytes, 
                                                // where N is the UTF-8 character length.
                                                // 11______ 0_______
                                                // 11______ 11______

    constexpr const uint8_t TOO_LONG    = 1<<1; // The leading byte must not be a continuation byte.
                                                // 0_______ 10______

    constexpr const uint8_t OVERLONG_2  = 1<<5; // Above U+7F for two-byte characters,
                                                // 1100000_ 10______
    constexpr const uint8_t OVERLONG_3  = 1<<2; // Above U+7FF for three-byte characters,
                                                // 11100000 100_____
    constexpr const uint8_t OVERLONG_4  = 1<<6; // Above U+7FFF for three-byte characters,
                                                // 11110000 1000____

    constexpr const uint8_t SURROGATE   = 1<<4; // The decoded character must be not be in U+D800...DFFF
                                                // 11101101 101_____

    constexpr const uint8_t TWO_CONTS   = 1<<7; // Two continious bit after each other
                                                // 10______ 10______

    constexpr const uint8_t TOO_LARGE   = 1<<3; // The decoded character must be less than or equal to U+10FFFF
                                                // 11110100 1001____
                                                // 11110100 101_____
                                                // 11110101 1001____
                                                // 11110101 101_____
                                                // 1111011_ 1001____
                                                // 1111011_ 101_____
                                                // 11111___ 1001____
                                                // 11111___ 101_____

    constexpr const uint8_t TOO_LARGE_1000 = 1<<6;
                                                // Out of the range, it must be maximum 100 if you see 0101, 011_, or 1___
                                                // 11110101 1000____
                                                // 1111011_ 1000____
                                                // 11111___ 1000____


    constexpr const uint8_t CARRY = TOO_SHORT | TOO_LONG | TWO_CONTS; 
                                                // These all have ____ in byte 1 . 10000011


    
    // SIMDJSON use table in CPU, but in GPU Table is very slow
    // we check 4 character in a single time by this:
    constexpr const uint32_t TOO_SHORT_32 = (
        ((uint32_t)TOO_SHORT)       | 
        ((uint32_t)TOO_SHORT) << 8  | 
        ((uint32_t)TOO_SHORT) << 16 | 
        ((uint32_t)TOO_SHORT) << 24
    );
    constexpr const uint32_t TOO_LONG_32 = (
        ((uint32_t)TOO_LONG)        | 
        ((uint32_t)TOO_LONG) << 8   |
        ((uint32_t)TOO_LONG) << 16  |
        ((uint32_t)TOO_LONG) << 24
    );
    constexpr const uint32_t OVERLONG_2_32 = (
        ((uint32_t)OVERLONG_2)       | 
        ((uint32_t)OVERLONG_2) << 8  | 
        ((uint32_t)OVERLONG_2) << 16 | 
        ((uint32_t)OVERLONG_2) << 24
    );
    constexpr const uint32_t OVERLONG_3_32 = (
        ((uint32_t)OVERLONG_3)       | 
        ((uint32_t)OVERLONG_3) << 8  | 
        ((uint32_t)OVERLONG_3) << 16 | 
        ((uint32_t)OVERLONG_3) << 24
    );
    constexpr const uint32_t OVERLONG_4_32 = (
        ((uint32_t)OVERLONG_4)       | 
        ((uint32_t)OVERLONG_4) << 8  | 
        ((uint32_t)OVERLONG_4) << 16 | 
        ((uint32_t)OVERLONG_4) << 24
    );
    constexpr const uint32_t SURROGATE_32 = (
        ((uint32_t)SURROGATE)       | 
        ((uint32_t)SURROGATE) << 8  | 
        ((uint32_t)SURROGATE) << 16 | 
        ((uint32_t)SURROGATE) << 24
    );
    constexpr const uint32_t TWO_CONTS_32 = (
        ((uint32_t)TWO_CONTS)       | 
        ((uint32_t)TWO_CONTS) << 8  | 
        ((uint32_t)TWO_CONTS) << 16 | 
        ((uint32_t)TWO_CONTS) << 24
    );
    constexpr const uint32_t TOO_LARGE_32 = (
        ((uint32_t)TOO_LARGE)       | 
        ((uint32_t)TOO_LARGE) << 8  | 
        ((uint32_t)TOO_LARGE) << 16 | 
        ((uint32_t)TOO_LARGE) << 24
    );
    constexpr const uint32_t TOO_LARGE_1000_32 = (
        ((uint32_t)TOO_LARGE_1000)       | 
        ((uint32_t)TOO_LARGE_1000) << 8  | 
        ((uint32_t)TOO_LARGE_1000) << 16 | 
        ((uint32_t)TOO_LARGE_1000) << 24
    );
    constexpr const uint32_t CARRY_32 = (
        ((uint32_t)CARRY)       | 
        ((uint32_t)CARRY) << 8  | 
        ((uint32_t)CARRY) << 16 |
        ((uint32_t)CARRY) << 24
    );
    



    // [2_high,  ] [1_high ,1_low] <--
    // [a,b,c,d,e,f,g,h] --shr-->   [0,0,0,0,a,b,c,d]
    //                              [0,0,0,0,1,0,0,0] {08}
    //--> kochak tr bashe -->       [1,1,1,1,1,1,1,1]
    //                              [0,0,0,0,0,0,1,0] {too long}

    uint32_t prev1_current = prev1;
    uint32_t byte_1 = 
        (__vcmpltu4(prev1_current, 0x80808080) & TOO_LONG_32) |
        (__vcmpgeu4(prev1_current, 0xC0C0C0C0) & TOO_SHORT_32) | 
        ( (__vcmpeq4(prev1_current, 0xC0C0C0C0) | __vcmpeq4(prev1_current, 0xC1C1C1C1)) & OVERLONG_2_32) | 
        (__vcmpeq4(prev1_current, 0xEDEDEDED) & (SURROGATE_32)) | 
        (__vcmpeq4(prev1_current, 0xE0E0E0E0) & (OVERLONG_3_32)) | 
        (__vcmpeq4(prev1_current, 0xF0F0F0F0) & (OVERLONG_4_32)) | 
        (__vcmpgtu4(prev1_current, 0xF4F4F4F4) & TOO_LARGE_1000_32) | 
        (__vcmpgtu4(prev1_current, 0xF3F3F3F3) & TOO_LARGE_32);
    byte_1 = 
        (__vcmpeq4(byte_1, 0x00000000) & TWO_CONTS_32);
        // (__vcmpgeu4(prev1_current, 0x80808080) & __vcmpltu4(prev1_current, 0xC0C0C0C0) & TWO_CONTS_32);
        // (__vcmpeq4(0x80808080 & prev1_current, 0x80808080) & TWO_CONTS_32);


    // __vcmpltu4 --> compare function in GPU: less than 
    // __vcmpgeu4 --> compare function in GPU: more or equal
    // work with shift right of prev1
    // uint32_t shr_prev1 = (prev1 >> 4) & 0x0f0f0f0f; // shift right --> extract high-order 4 bitgifaz
    // uint32_t byte_1_high = // Byte 1 (8 bit - 1 Character [ascii])--> 4 khone bala
    //     (__vcmpltu4(shr_prev1, 0x08080808) & TOO_LONG_32) | 
    //     (__vcmpgeu4(shr_prev1, 0x08080808) & __vcmpltu4(shr_prev1, 0x0C0C0C0C) & TWO_CONTS_32) | 
    //     (__vcmpgeu4(shr_prev1, 0x0C0C0C0C) & TOO_SHORT_32) | 
    //     (__vcmpeq4 (shr_prev1, 0x0C0C0C0C) & OVERLONG_2_32) | 
    //     (__vcmpeq4 (shr_prev1, 0x0E0E0E0E) & (OVERLONG_3_32 | SURROGATE_32)) | 
    //     (__vcmpeq4 (shr_prev1, 0x0F0F0F0F) & (TOO_LARGE_32 | TOO_LARGE_1000_32 | OVERLONG_4_32));


    // work with shift left of prev1
    // uint32_t shl_prev1 = prev1 & 0x0f0f0f0f;        
    // uint32_t byte_1_low =  // Byte 1 (8 bit - 1 Character[ascii])--> 4 khone paeen
    //     (CARRY_32) | 
    //     (__vcmpltu4(shl_prev1, 0x02020202) & OVERLONG_2_32) |
    //     (__vcmpgeu4(shl_prev1, 0x04040404) & TOO_LARGE_32) | 
    //     (__vcmpgtu4(shl_prev1, 0x04040404) & TOO_LARGE_1000_32) | 
    //     (__vcmpeq4 (shl_prev1, 0) & (OVERLONG_3_32 | OVERLONG_4_32)) | 
    //     (__vcmpeq4 (shl_prev1, 0x0D0D0D0D) & SURROGATE_32);
    

    uint32_t block_compressed_high = (block_compressed >> 4) & 0x0F0F0F0F; 
    // 4 khune bala ro brdshti 
    // baraye moqaysee adadi bordim daste rast k rahat tr bashe


    // to make it more easier than before, save it and use it multiple time
    uint32_t less_than_12 = __vcmpltu4(block_compressed_high, 0x0C0C0C0C);
    uint32_t byte_2_high = 
        ((__vcmpltu4(block_compressed_high, 0x08080808) | __vcmpgtu4(block_compressed_high, 0x0B0B0B0B)) & TOO_SHORT_32) |
        (less_than_12 & __vcmpgeu4(block_compressed_high, 0x08080808) & (TOO_LONG_32 | OVERLONG_2_32 | TWO_CONTS_32)) | 
        (less_than_12 & __vcmpgtu4(block_compressed_high, 0x08080808) & TOO_LARGE_32) | 
        (__vcmpeq4(block_compressed_high, 0x08080808) & (TOO_LARGE_1000_32 | OVERLONG_4_32)) | 
        (__vcmpgtu4(block_compressed_high, 0x09090909) & less_than_12 & SURROGATE_32); 


    // result =   (byte_1_high & byte_1_low & byte_2_high); 
    result =   (byte_1 & byte_2_high); 
    
    // 0 --> okay and return secussfuly
}

// make sure it has 2 or 3 continuation
// for 3,4 Byte
__device__ __forceinline__
void continuationBytes(uint32_t prev2, uint32_t prev3, uint32_t sc, uint32_t& must32Upper_sc, uint64_t size, int total_padded_32){
    static const uint32_t third_subtract_byte =  
    // 11100000 - 1 --> 11011111 --> This is the maximum of 2 Byte, So if it’s more than this, we have 3 
        (0b11100000u-1)       | 
        (0b11100000u-1) << 8  | 
        (0b11100000u-1) << 16 | 
        (0b11100000u-1) << 24;

    static const uint32_t fourth_subtract_byte = 
        (0b11110000u-1)       | 
        (0b11110000u-1) << 8  |
        (0b11110000u-1) << 16 | 
        (0b11110000u-1) << 24;


    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // the latest byte in our UTF8Bytes (character) is third or fourth
    // subtract prev2 and prev3 from third_subtract_byte and fourth_subtract_byte
    // must be 0 
    // unsign saturated subtraction 4 Byte --> 4 Byte ro parallel az ham kam mikone ya 0 mishe ya 1 
    // ma mikhaym prev2 az third_subtract_byte va prev3 az forth_subtract_byte kochak tr bashe k javab 0 bashe
    uint32_t is_third_byte  = __vsubus4(prev2, third_subtract_byte);
    uint32_t is_fourth_byte = __vsubus4(prev3, fourth_subtract_byte);


    uint32_t gt = ( __vsubss4((int32_t)(is_third_byte | is_fourth_byte), int32_t(0)) ) & 0xFFFFFFFF; 
    
    // because we are working in 32 bit, we need do this for all 4 characters
    uint32_t must32 = __vcmpgtu4(gt, 0); // gt --> hamin must32 hast o mitonim hazfesh knim

    must32Upper_sc = (must32 & 0x80808080) ^ sc;            //  sc --> output of 32 bit check
    // upper bit of each 4 character
} 

__global__ 
void checkAscii(uint32_t* blockCompressed_GPU, uint64_t size, int total_padded_32, bool* hastUTF8, int WORDS){
    int threadId = threadIdx.x;
    __shared__ uint32_t shared_flag;
    
    if(threadId == 0) shared_flag = 0;
    __syncthreads();

    int index = blockIdx.x * blockDim.x + threadId;
    int stride = blockDim.x * gridDim.x;

    for(long i = index; i< total_padded_32; i+=stride){
        int start = i*WORDS;
        #pragma unroll
        for(int j=start; j<size && j<start+WORDS; j++){
            if((blockCompressed_GPU[j] & 0x80808080) != 0) atomicOr(&shared_flag, 1); 
            // check the upper bit
            // atomic or because it works in parallel
        }
        __syncthreads();
    }
    if(threadId == 0 && shared_flag) *hastUTF8 = true;
}

__global__
void checkUTF8(uint32_t* blockCompressed_GPU, uint32_t* error_GPU, uint64_t size, int total_padded_32, int WORDS){
    /*
    - blockCompressed_GPU is a pointer to the compressed data block in GPU memory, 
    - error_GPU is a pointer to a location in GPU memory where the function will store an error code if it detects invalid UTF-8, 
    - size is the size of the data block, 
    - total_padded_32 is the total number of 32-bit words in the padded data block, and 
    - WORDS is the number of words processed by each thread in each iteration of the loop
    */
    static const uint32_t max_val = 
        (uint32_t)(0b11000000u-1 << 24) | 
        (uint32_t)(0b11100000u-1 << 16) | 
        (uint32_t)(0b11110000u-1 << 8)  | 
        (uint32_t)(255); 

    int threadId = threadIdx.x;
    __shared__ uint32_t shared_error;
    if(threadId == 0) shared_error = 0;

    __syncthreads();
    int index = blockIdx.x * blockDim.x + threadId;
    int stride = blockDim.x * gridDim.x;

    for(long i = index; i< total_padded_32; i+=stride){
        int start = i*WORDS;
        #pragma unroll
        for(int j=start; j<size && j<start+WORDS; j++){
            uint32_t current = blockCompressed_GPU[j];
            uint32_t previous = j>0 ? blockCompressed_GPU[j-1] : 0;
            uint32_t prev_incomplete = __vsubus4(previous, max_val);
            
            if((current & 0x80808080) == 0) {
                atomicExch(&shared_error, prev_incomplete);
            }else{
                uint32_t prev1, prev2, prev3;
                uint32_t sc;
                uint32_t must32Upper_sc;

                uint64_t dist = ( ((uint64_t)current) << 32) | (uint64_t) previous;
                prev1 = (uint32_t)(dist >> 3*8); // shifted by 3 byte (3 * 8 bits)
                prev2 = (uint32_t)(dist >> 2*8); // shifted by 2 byte (2 * 8 bits)
                prev3 = (uint32_t)(dist >> 1*8); // shifted by 1 byte (1 * 8 bits)

                vectorizedClassification(current, prev1, sc, size, total_padded_32); // check 1,2 Byte 
                continuationBytes(prev2, prev3, sc, must32Upper_sc, size, total_padded_32); // Check 3,4 byte

                atomicExch(&shared_error, must32Upper_sc); // return error
            }
        }
    }
    __syncthreads();
    if(threadId==0 && shared_error) *error_GPU = shared_error;
}

inline bool UTF8Validation(uint32_t * block_GPU, uint64_t size){
    // _________________INIT_________________________
    int total_padded_32 = size;

    uint32_t* general_ptr;
    hipMallocAsync(&general_ptr, sizeof(uint32_t), 0);
    uint32_t* error_GPU = general_ptr;
    hipMemsetAsync(error_GPU, 0, sizeof(uint32_t), 0);

  
    int total_padded_16B = (size+3)/4;
    int WORDS = 4;
    int numBlock_16B = (total_padded_16B+BLOCKSIZE-1) / BLOCKSIZE;


    bool hastUTF8 = false;
    bool* hastUTF8_GPU;
    hipMallocAsync(&hastUTF8_GPU, sizeof(bool), 0);                  //  Allocates Memory on the Device and Returns a Pointer to the Allocated Memory.
    hipMemsetAsync(hastUTF8_GPU, 0, sizeof(bool), 0);                //  Initializes a Block of Memory on the Device with a Specified Value
  
    //cout << "Validation Start:\n";
    // Prepare
    //t hipEvent_t start, stop;
    //t hipEventCreate(&start);
    //t hipEventCreate(&stop);
    // Start record
    //t hipEventRecord(start, 0);

    // _________________PART_1_______________________
    checkAscii<<<numBlock_16B, BLOCKSIZE>>>(block_GPU, size, total_padded_16B, hastUTF8_GPU, WORDS);
    hipStreamSynchronize(0);
    
    hipMemcpyAsync(&hastUTF8, hastUTF8_GPU, sizeof(bool), hipMemcpyDeviceToHost, 0);
    //hipFreeAsync(hastUTF8_GPU, 0);

    if(!hastUTF8){ 
        //printf("There is no utf8 charachter\n");
        hipFreeAsync(general_ptr, 0);
        //hipFreeAsync(hastUTF8_GPU, 0);
        return true;
    }


    // _________________PART_2_______________________
    checkUTF8<<<numBlock_16B, BLOCKSIZE>>>(block_GPU, error_GPU, size, total_padded_16B, WORDS);
    hipStreamSynchronize(0);

    // _________________RESULT_______________________
    // Stop event
    //t hipEventRecord(stop, 0);
    //t hipEventSynchronize(stop);
    //t float elapsedTime;
    //t hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    //t hipEventDestroy(start);
    //t hipEventDestroy(stop);
    //t time_cal.validation_t += elapsedTime;

    uint32_t error = 0;
    hipMemcpyAsync(&error, error_GPU, sizeof(uint32_t), hipMemcpyDeviceToHost, 0);
    hipFreeAsync(general_ptr, 0);
    if(error != 0){ 
        printf("Incomplete ASCII!\n"); 
        //hipFreeAsync(error_GPU, 0);
        //hipFreeAsync(hastUTF8_GPU, 0);
        return false;
    }
            
    //udaFreeAsync(error_GPU, 0);
    //hipFreeAsync(hastUTF8_GPU, 0);
    return true;

}

__global__
void bitMapCreator(uint8_t* block_GPU, uint32_t* outputSlash, uint32_t* outputQuote, uint32_t* op_GPU, uint32_t* newLine_GPU, uint64_t size, int total_padded_32){
    /*
    The purpose of this function is to create bitmaps for different characters in the data block. 
    Each bitmap represents the presence or absence of a specific character at a particular position in the data block.

    For Example:
    Let's say the data block is {"name": "John\/Doe"}. 
        {"name":"John\/Doe"}
    \:  00000000000000100000
    ":  01000010100000000010
    op: 10000001000000000001

    
    Also, let's assume that the data block has a total of 64 characters (so size = 64) and 
    total_padded_32 = 2 (assuming we have 2 32-bit elements after padding).
    The resulting bitmap will be stored in the outputSlash array. 
    The value of output1[0] will represent the bitmap for the first 32 characters, 
    and output1[1] will represent the bitmap for the remaining 32 characters.
    */


    /*
    - block_GPU: A pointer to an array of uint8_t (unsigned 8-bit integers), representing a block of data.
    - outputSlash, outputQuote, op_GPU, and newLine_GPU: Pointers to arrays of uint32_t (unsigned 32-bit integers), which will store the output results.
    - size: A uint64_t (unsigned 64-bit integer) representing the size of the data block.
    - total_padded_32: An int representing the total number of 32-bit elements after padding.
    */


    /*
    The variables blockIdx.x, blockDim.x, and threadIdx.x are used to calculate the current thread's index
    and the stride value (stride) that determines the loop iterations for each thread.
    */
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;


    for(long i = index; i< total_padded_32; i+=stride){
        // loop inside data among blocks
        // The for loop iterates over the range of total_padded_32 with a step size of stride. 
        // This loop distributes the work among different threads to process different parts of the data block.
        int start_position = i*32;
        // the variable start_position is calculated based on the current index (i) to 
        // determine the starting position of the 32-bit segment being processed.

        // These variables will store the bitmaps for different characters:    
        uint32_t res_slash = 0;     //  " / "
        uint32_t res_quote = 0;     //  " " "
        uint32_t res_op = 0;        //  " { } [ ] : ,"
        uint32_t res_newline = 0;   //  " \n "

        for (int j = start_position; j<start_position+32 && j<size; j++){
            // Loop inside blocks between threads
            // Another nested loop iterates over the range from start to start+32 and 
            // ensures that the loop doesn't go beyond the size of the data block (size).
            uint8_t block = block_GPU[j];
            // uint8_t block_low = block & 0x08; // This operation isolates the 4th bit of the block value.

            // |= : bitwise OR operation 
            // << :bitwise SHIF operation
            block == '\\' ? res_slash |= 1 << (j-start_position) : NULL; // or-shif: first or then shift 
            block == '\"' ? res_quote |= 1 << (j-start_position) : NULL;
            block == '\n' ? res_newline |= 1 << (j-start_position): NULL;
            res_op |= ((( //operands
                    block == '{' ||
                    block == '[' ||
                    block == '}' ||
                    block == ']' ||
                    block == ':' ||
                    block == ','
                    ) ? 1 : 0) << (j-start_position)) ;
            
            // res_newline |= ((( // new line
            //         //block == ' ' ||
            //         //block == '\t' ||
            //         //block == '\r'
            //         block == '\n'
            //         ) ? 1 : 0) << (j-start_position)) ;

    
        }

        // creating bit-map for this 4 results-->
        outputSlash[i] = res_slash;      // " \ "
        outputQuote[i] = res_quote;      // " " "
        op_GPU[i] = res_op;           // operands
        newLine_GPU[i] = res_newline;   // \n
    }

    /*
    Example of how it works:
    Suppose we have the following data block: 
        "Hello\\World".
    We want to create a bitmap to represent the backslash character '\\' in the data block.
    Let's assume that we are currently processing the 10th character in the data block, and (j-start_position) is 10.

    Now, when we reach the 10th character, which is the backslash '\\', the condition block == '\\' will evaluate to true.
    So, the expression res1 |= 1 << (j-start_position) will be executed. 
    Since j-start is 10 in this example, 
        the bitwise left shift operation 1 << (j-start) will shift the number 1 by 10 positions to the left, 
        resulting in the binary number 0b00000000001.

    Then, the bitwise OR operation res_slash |= ... will be performed. 
    Suppose the initial value of res_slash is 0. The operation res1 |= 0b00000000001 will update res1 by setting the 10th bit to 1.
    
    After processing the entire data block, 
    the resulting res_slash bitmap will represent the presence or absence of the backslash character '\\' at each position.

    In the following character if we found any other "\\", 
        the result of this approach again would be the seeting the (j-size_position)th bit to 1.  
    */
}


//     bitMapCreatorSimd<<<numBlock_8, BLOCKSIZE>>>( (uint32_t*) block_GPU, (uint8_t*) backslashes_GPU, (uint8_t*) quote_GPU, (uint8_t*) op_GPU, (uint8_t*) open_close_GPU, size, total_padded_8);

__global__
void bitMapCreatorSimd(uint32_t* block_GPU, uint8_t* outputSlash, uint8_t* outputQuote, uint8_t* op_GPU, uint8_t* open_close_GPU, uint64_t size, int total_padded_8){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = index; i < total_padded_8 && i < size; i += stride) {
        
        int start = i*2;
        
        // if (start < total_padded_8) {
        //     printf("GPU index-%d --> %d\n", start, block_GPU[start]);
        // }
        uint8_t res_slash = 0;     //  " / "
        uint8_t res_quote = 0;     //  " " "
        uint8_t res_op = 0;        //  " { } [ ] : ,"
        uint8_t res_open_close = 0;   //  " \n "

        uint32_t temp_res_slash = 0;
        uint32_t temp_res_quote = 0;
        uint32_t temp_res_op = 0;
        uint32_t temp_colon_comma_newline = 0;
        uint32_t temp_open_close = 0;


        uint32_t block = block_GPU[start];
        // printf("index-%d --> %d\n", i, block_GPU[i]);

        temp_res_slash = (__vcmpeq4(block, 0x5C5C5C5C) & 0x01010101); // 00000000 00000001 00000001 00000001
        temp_res_quote = (__vcmpeq4(block, 0x22222222) & 0x01010101);
        temp_open_close = ((
                    __vcmpeq4(block, 0x5B5B5B5B) |
                    __vcmpeq4(block, 0x5D5D5D5D) |
                    __vcmpeq4(block, 0x7B7B7B7B) |
                    __vcmpeq4(block, 0x7D7D7D7D) ) & 0x01010101);

        temp_colon_comma_newline = ((
                    __vcmpeq4(block, 0x3A3A3A3A) |
                    __vcmpeq4(block, 0x2C2C2C2C) |
                    __vcmpeq4(block, 0x0A0A0A0A)) & 0x01010101);

        temp_res_op = temp_colon_comma_newline | temp_open_close;
        // temp_res_newline = (__vcmpeq4(block, 0x32323232) & 0x01010101);
    
        
        // int size_4 = (size + 3)/4;
        if(i == total_padded_8 - 1 && 4*(start + 1)  >= size ){
            // ((uint32_t*) outputSlash)[start] = 0;
            // ((uint32_t*) outputQuote)[start] = 0;
            // ((uint32_t*) op_GPU)[start] = 0;
            // ((uint32_t*) open_close_GPU)[start] = 0;

            for(int j = 0; j < 4; j++){
                res_slash   |= (uint8_t) (temp_res_slash >> j*7 & 0x0F) ;
                res_quote   |= (uint8_t) (temp_res_quote >> j*7 & 0x0F);
                res_op      |= (uint8_t) (temp_res_op >> j*7 & 0x0F   );
                res_open_close |= (uint8_t) (temp_open_close >> j*7 & 0x0F );
            }
            outputSlash[i] = res_slash;      // " \ "
            outputQuote[i] = res_quote;      // " " "
            op_GPU[i] = res_op;              // operands
            open_close_GPU[i] = res_open_close;    // \n
            continue;
        }

        uint32_t temp2_res_slash = 0;
        uint32_t temp2_res_quote = 0;
        uint32_t temp2_res_op = 0;
        uint32_t temp2_colon_comma_newline = 0;
        uint32_t temp2_open_close= 0;


      

        uint32_t block_2 = block_GPU[start+1];
        // printf("index-%d -2-> %d\n", i, block_2);
        temp2_res_slash = (__vcmpeq4(block_2, 0x5C5C5C5C) & 0x01010101); // 00000001 00000001 00000001 00000001
        temp2_res_quote = (__vcmpeq4(block_2, 0x22222222) & 0x01010101);

        temp2_open_close = (
                    __vcmpeq4(block_2, 0x5B5B5B5B) |
                    __vcmpeq4(block_2, 0x5D5D5D5D) |
                    __vcmpeq4(block_2, 0x7B7B7B7B) |
                    __vcmpeq4(block_2, 0x7D7D7D7D) ) & 0x01010101;

        temp2_colon_comma_newline = (
                    __vcmpeq4(block_2, 0x3A3A3A3A) |
                    __vcmpeq4(block_2, 0x2C2C2C2C) |
                    __vcmpeq4(block_2, 0x0A0A0A0A)) & 0x01010101;

        temp2_res_op = temp2_colon_comma_newline | temp2_open_close;


        for(int j = 0; j < 4; j++){
            //   j=0     00000001               | 00000001 << 3 = 00010000
            //   j=1     00000010               | 00000010 << 3
            //   j=2     00000100               | 00000100 << 3
            //   j=3     ...
            res_slash   |= (uint8_t) ((temp_res_slash >> j*7) | ((temp2_res_slash >> j*7) << 4) );
            res_quote   |= (uint8_t) (temp_res_quote >> j*7 | ((temp2_res_quote >> j*7) << 4) );
            res_op      |= (uint8_t) (temp_res_op >> j*7    | ((temp2_res_op >> j*7) << 4) );
            res_open_close |= (uint8_t) (temp_open_close >> j*7 | ((temp2_open_close >> j*7) << 4) );
        }


        // if(index == 15) {
        //     char binary1[33], binary2[33];
        //     u32ToBinary(temp_res_quote, binary1);
        //     u32ToBinary(temp2_res_quote, binary2);
        //     printf("temp1: %s\n", binary1);
        //     printf("temp2: %s\n", binary2);
        //     printf("merge: %s\n", byteToBinary(res_quote));
        // }
        // 0-1: 0
        // 2-3: 1
        // 4-5: 2
        // 6-7: 3
        // cout << "index-" << i << "-->" << res_quote << endl;
        // printf("index-%d --> %d\n", i, res_quote);
        // storing results in the larger arrays
        outputSlash[i] = res_slash;      // " \ "
        outputQuote[i] = res_quote;      // " " "
        op_GPU[i] = res_op;              // operands
        open_close_GPU[i] = res_open_close;    // \n
    }
}


// must change based on bitMapCreatorSimd
__global__
void bitMapCreatorSimd32(uint32_t* block_GPU, uint32_t* outputSlash, uint32_t* outputQuote, uint32_t* op_GPU, uint32_t* newLine_GPU, uint64_t size, int total_padded_32){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = index; i < total_padded_32 && i < size; i += stride) {
        int start = i*8;
        // if (start < total_padded_32) {
        //     printf("GPU index-%d --> %d\n", start, block_GPU[start]);
        // }
        uint32_t res_slash = 0;     //  " / "
        uint32_t res_quote = 0;     //  " " "
        uint32_t res_op = 0;        //  " { } [ ] : ,"
        uint32_t res_newline = 0;   //  " \n "

        uint32_t temp_res_slash = 0;
        uint32_t temp_res_quote = 0;
        uint32_t temp_res_op = 0;
        uint32_t temp_res_newline = 0;

        uint32_t block = block_GPU[start];
        // printf("index-%d --> %d\n", i, block_GPU[i]);

        temp_res_slash = (__vcmpeq4(block, 0x5C5C5C5C) & 0x01010101); // 00000000 00000001 00000001 00000001
        temp_res_quote = (__vcmpeq4(block, 0x22222222) & 0x01010101);
        temp_res_op = ((__vcmpeq4(block, 0x5B5B5B5B) |
                    __vcmpeq4(block, 0x5D5D5D5D) |
                    __vcmpeq4(block, 0x7B7B7B7B) |
                    __vcmpeq4(block, 0x7D7D7D7D) |
                    __vcmpeq4(block, 0x3A3A3A3A) |
                    __vcmpeq4(block, 0x2C2C2C2C)) & 0x01010101);
        temp_res_newline = (__vcmpeq4(block, 0x32323232) & 0x01010101);

        uint32_t temp2_res_slash = 0;
        uint32_t temp2_res_quote = 0;
        uint32_t temp2_res_op = 0;
        uint32_t temp2_res_newline= 0;

        uint32_t block_2 = block_GPU[start+1];
        // printf("index-%d -2-> %d\n", i, block_2);
        temp2_res_slash = (__vcmpeq4(block_2, 0x5C5C5C5C) & 0x01010101); // 00000001 00000001 00000001 00000001
        temp2_res_quote = (__vcmpeq4(block_2, 0x22222222) & 0x01010101);
        temp2_res_op = ((__vcmpeq4(block_2, 0x5B5B5B5B) |
                    __vcmpeq4(block_2, 0x5D5D5D5D) |
                    __vcmpeq4(block_2, 0x7B7B7B7B) |
                    __vcmpeq4(block_2, 0x7D7D7D7D) |
                    __vcmpeq4(block_2, 0x3A3A3A3A) |
                    __vcmpeq4(block_2, 0x2C2C2C2C)) & 0x01010101);
        temp2_res_newline = (__vcmpeq4(block_2, 0x32323232) & 0x01010101);

        uint32_t temp3_res_slash = 0;
        uint32_t temp3_res_quote = 0;
        uint32_t temp3_res_op = 0;
        uint32_t temp3_res_newline = 0;

        uint32_t block_3 = block_GPU[start+2];
        // printf("index-%d --> %d\n", i, block_GPU[i]);

        temp3_res_slash = (__vcmpeq4(block_3, 0x5C5C5C5C) & 0x01010101); // 00000000 00000001 00000001 00000001
        temp3_res_quote = (__vcmpeq4(block_3, 0x22222222) & 0x01010101);
        temp3_res_op = ((__vcmpeq4(block_3, 0x5B5B5B5B) |
                    __vcmpeq4(block_3, 0x5D5D5D5D) |
                    __vcmpeq4(block_3, 0x7B7B7B7B) |
                    __vcmpeq4(block_3, 0x7D7D7D7D) |
                    __vcmpeq4(block_3, 0x3A3A3A3A) |
                    __vcmpeq4(block_3, 0x2C2C2C2C)) & 0x01010101);
        temp3_res_newline = (__vcmpeq4(block_3, 0x32323232) & 0x01010101);

        uint32_t temp4_res_slash = 0;
        uint32_t temp4_res_quote = 0;
        uint32_t temp4_res_op = 0;
        uint32_t temp4_res_newline = 0;

        uint32_t block_4 = block_GPU[start+3];
        // printf("index-%d --> %d\n", i, block_GPU[i]);

        temp4_res_slash = (__vcmpeq4(block_4, 0x5C5C5C5C) & 0x01010101); // 00000000 00000001 00000001 00000001
        temp4_res_quote = (__vcmpeq4(block_4, 0x22222222) & 0x01010101);
        temp4_res_op = ((__vcmpeq4(block_4, 0x5B5B5B5B) |
                    __vcmpeq4(block_4, 0x5D5D5D5D) |
                    __vcmpeq4(block_4, 0x7B7B7B7B) |
                    __vcmpeq4(block_4, 0x7D7D7D7D) |
                    __vcmpeq4(block_4, 0x3A3A3A3A) |
                    __vcmpeq4(block_4, 0x2C2C2C2C)) & 0x01010101);
        temp4_res_newline = (__vcmpeq4(block_4, 0x32323232) & 0x01010101);

        uint32_t temp5_res_slash = 0;
        uint32_t temp5_res_quote = 0;
        uint32_t temp5_res_op = 0;
        uint32_t temp5_res_newline= 0;

        uint32_t block_5 = block_GPU[start+4];
        // printf("index-%d -2-> %d\n", i, block_2);
        temp5_res_slash = (__vcmpeq4(block_5, 0x5C5C5C5C) & 0x01010101); // 00000001 00000001 00000001 00000001
        temp5_res_quote = (__vcmpeq4(block_5, 0x22222222) & 0x01010101);
        temp5_res_op = ((__vcmpeq4(block_5, 0x5B5B5B5B) |
                    __vcmpeq4(block_5, 0x5D5D5D5D) |
                    __vcmpeq4(block_5, 0x7B7B7B7B) |
                    __vcmpeq4(block_5, 0x7D7D7D7D) |
                    __vcmpeq4(block_5, 0x3A3A3A3A) |
                    __vcmpeq4(block_5, 0x2C2C2C2C)) & 0x01010101);
        temp5_res_newline = (__vcmpeq4(block_5, 0x32323232) & 0x01010101);

        uint32_t temp6_res_slash = 0;
        uint32_t temp6_res_quote = 0;
        uint32_t temp6_res_op = 0;
        uint32_t temp6_res_newline = 0;

        uint32_t block_6 = block_GPU[start+5];
        // printf("index-%d --> %d\n", i, block_GPU[i]);

        temp6_res_slash = (__vcmpeq4(block_6, 0x5C5C5C5C) & 0x01010101); // 00000000 00000001 00000001 00000001
        temp6_res_quote = (__vcmpeq4(block_6, 0x22222222) & 0x01010101);
        temp6_res_op = ((__vcmpeq4(block_6, 0x5B5B5B5B) |
                    __vcmpeq4(block_6, 0x5D5D5D5D) |
                    __vcmpeq4(block_6, 0x7B7B7B7B) |
                    __vcmpeq4(block_6, 0x7D7D7D7D) |
                    __vcmpeq4(block_6, 0x3A3A3A3A) |
                    __vcmpeq4(block_6, 0x2C2C2C2C)) & 0x01010101);
        temp6_res_newline = (__vcmpeq4(block_6, 0x32323232) & 0x01010101);

        uint32_t temp7_res_slash = 0;
        uint32_t temp7_res_quote = 0;
        uint32_t temp7_res_op = 0;
        uint32_t temp7_res_newline= 0;

        uint32_t block_7 = block_GPU[start+6];
        // printf("index-%d -2-> %d\n", i, block_2);
        temp7_res_slash = (__vcmpeq4(block_7, 0x5C5C5C5C) & 0x01010101); // 00000001 00000001 00000001 00000001
        temp7_res_quote = (__vcmpeq4(block_7, 0x22222222) & 0x01010101);
        temp7_res_op = ((__vcmpeq4(block_7, 0x5B5B5B5B) |
                    __vcmpeq4(block_7, 0x5D5D5D5D) |
                    __vcmpeq4(block_7, 0x7B7B7B7B) |
                    __vcmpeq4(block_7, 0x7D7D7D7D) |
                    __vcmpeq4(block_7, 0x3A3A3A3A) |
                    __vcmpeq4(block_7, 0x2C2C2C2C)) & 0x01010101);
        temp7_res_newline = (__vcmpeq4(block_7, 0x32323232) & 0x01010101);


        uint32_t temp8_res_slash = 0;
        uint32_t temp8_res_quote = 0;
        uint32_t temp8_res_op = 0;
        uint32_t temp8_res_newline= 0;

        uint32_t block_8 = block_GPU[start+7];
        // printf("index-%d -2-> %d\n", i, block_2);
        temp8_res_slash = (__vcmpeq4(block_8, 0x5C5C5C5C) & 0x01010101); // 00000001 00000001 00000001 00000001
        temp8_res_quote = (__vcmpeq4(block_8, 0x22222222) & 0x01010101);
        temp8_res_op = ((__vcmpeq4(block_8, 0x5B5B5B5B) |
                    __vcmpeq4(block_8, 0x5D5D5D5D) |
                    __vcmpeq4(block_8, 0x7B7B7B7B) |
                    __vcmpeq4(block_8, 0x7D7D7D7D) |
                    __vcmpeq4(block_8, 0x3A3A3A3A) |
                    __vcmpeq4(block_8, 0x2C2C2C2C)) & 0x01010101);
        temp8_res_newline = (__vcmpeq4(block_8, 0x32323232) & 0x01010101);

        for(int j = 0; j < 4; j++){
            //   j=0     00000001               | 00000001 << 3 = 00010000
            //   j=1     00000010               | 00000010 << 3
            //   j=2     00000100               | 00000100 << 3
            //   j=3     ...
            res_slash   |= ((temp_res_slash >> j*7)          | ((temp2_res_slash >> j*7) << 4) | 
                            ((temp3_res_slash >> j*7) << 8)  | ((temp4_res_slash >> j*7) << 12) |
                            ((temp5_res_slash >> j*7) << 16)  | ((temp6_res_slash >> j*7) << 20) |
                            ((temp7_res_slash >> j*7) << 24) | ((temp8_res_slash >> j*7) << 28) 
                            );
            res_quote   |= ((temp_res_quote >> j*7)          | ((temp2_res_quote >> j*7) << 4) | 
                            ((temp3_res_quote >> j*7) << 8)  | ((temp4_res_quote >> j*7) << 12) |
                            ((temp5_res_quote >> j*7) << 16)  | ((temp6_res_quote >> j*7) << 20) |
                            ((temp7_res_quote >> j*7) << 24) | ((temp8_res_quote >> j*7) << 28) 
                            );
            res_op      |= ((temp_res_op >> j*7)          | ((temp2_res_op >> j*7) << 4) | 
                            ((temp3_res_op >> j*7) << 8)  | ((temp4_res_op >> j*7) << 12) |
                            ((temp5_res_op >> j*7) << 16) | ((temp6_res_op >> j*7) << 20) |
                            ((temp7_res_op >> j*7) << 24) | ((temp8_res_op >> j*7) << 28) 
                            );
            res_newline |= ((temp_res_newline >> j*7)          | ((temp2_res_newline >> j*7) << 4) | 
                            ((temp3_res_newline >> j*7) << 8)  | ((temp4_res_newline >> j*7) << 12) |
                            ((temp5_res_newline >> j*7) << 16) | ((temp6_res_newline >> j*7) << 20) |
                            ((temp7_res_newline >> j*7) << 24) | ((temp8_res_newline >> j*7) << 28) 
                            );
        }


        outputSlash[i] = res_slash;      // " \ "
        outputQuote[i] = res_quote;      // " " "
        op_GPU[i] = res_op;              // operands
        newLine_GPU[i] = res_newline;    // \n
    }
}

__global__
void findEscapedQuoteMerge_NEW(uint32_t* backslashes_GPU, uint32_t* quote_GPU, uint32_t* real_quote_GPU, int size, int total_padded_32, int WORDS){
    /*
        The findEscapedQuote function analyzes the input data block and identifies the escaped characters. 
        It processes the data in parallel, utilizing bitwise operations to detect escape sequences 
        and mark the positions of non-escaped characters. 
        The resulting information is stored in the real_quote_GPU array for further processing or analysis.
    */


    // odd-length sequences of backslashes means we have escape character


    // OVERFLOW IS CAME FROM HIGH OF PREVIOUS WORD TO LOW BIT OF CURRENT WORD
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(long i = index; i< total_padded_32; i+=stride){
        int start = i*WORDS;

        // Parallel-For in GPU: 
        #pragma unroll
        for(int k=start; k<size && k<start+WORDS; k++){
            uint32_t overflow = 2;
            // It is used in combination with bitwise operations to detect --> 2 mean maybe overflow maybe not

            uint32_t evenBits = 0x55555555UL; // 5 --> 0101
            uint32_t oddBits = ~evenBits;
            // This is a uint32_t constant with a value of 0x55555555UL. 
            // It represents a bitmask with 1s in all even bit positions. 

            long j=k-1;
            if(k == 0) overflow = 0;
            uint32_t current_word_quote = quote_GPU[k];
            uint32_t backslashes = backslashes_GPU[k];                          //[0,1,1,1,0,0,0,1]

            uint32_t possible_escaped_quote =  current_word_quote & (backslashes << 1 | 1);  
            // this one is for finding possible escape double qutoes that we have to check
            if(possible_escaped_quote == 0){
                real_quote_GPU[k] = current_word_quote;
                quote_GPU[k] = (uint32_t) __popc(real_quote_GPU[k]);  // quote is total_one, we will rename it
                continue;
            }

            while(overflow == 2){
                uint32_t backslash_j = backslashes_GPU[j];                              //[1,1,1,0,0,0,0,0]
                // This is a uint32_t variable that stores the value of backslashes_GPU[j]. It represents the backslashes at position j in the input data.
                uint8_t following_backslash_counts = __clz(~backslash_j); // Convert to 0-based index
                overflow = (following_backslash_counts == 32) ? 2 : following_backslash_counts & 1; 
                j--; // previous chunk qable 
            }

            // has overflow at this step: 0 or 1
            // as same as SIMDJSON
            backslashes = backslashes & (~overflow);                            //[0,1,1,1,0,0,0,0] 
            uint32_t applyEscapedChar = (backslashes << 1) | overflow;            //[1,1,1,0,0,0,0,1] --> chn amaln yek backslash moaser bode k khonsa mikrde miomde to 

            // All BACKSLASHES that are at ODD LOCATION and not ESCAPED
            uint32_t oddSequence = backslashes & oddBits & ~applyEscapedChar;      
            uint32_t sequenceStartatEven = oddSequence + backslashes;           //[0,1,1,1,0,0,0,0]
            uint32_t invert_mask = sequenceStartatEven << 1;            //[1,1,1,0,0,0,0,0]
            uint32_t escaped = (evenBits ^ invert_mask) & applyEscapedChar;
            
            
            real_quote_GPU[k] = (~escaped) & current_word_quote;    // quote hae vaghie   
            quote_GPU[k] = (uint32_t) __popc(real_quote_GPU[k]);  // quote is total_one, we will rename it
        }
    }
}

// 1 WORD - Step 3:
// 1 Word + popc - scatter
__global__
void reduceChunkBaseline(uint32_t* real_quote_GPU, uint32_t* prediction_GPU, int total_padded_32){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for(long i = index; i < total_padded_32; i+=stride){
        prediction_GPU[i] = (uint32_t) __popc(real_quote_GPU[i]);
    }
}

// CUDA kernel where prefix_xor is called
// inStringFinderBaseline<<<numBlock, BLOCKSIZE>>>(real_quote_GPU, total_one_GPU, inString_GPU, total_padded_32);
__global__
void inStringFinderBaseline(uint32_t* real_quote_GPU, uint32_t* prefix_sum_ones, uint32_t* res, int total_padded_32){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < total_padded_32; i += stride){
        // Check if the prefix sum is odd (overflow)
        bool overflow = prefix_sum_ones[i] & 1;
        // Apply prefix_xor to real_quote_GPU
        res[i] = prefix_xor(real_quote_GPU[i]);
        // Update res[i] based on the overflow
        res[i] = overflow ? ~res[i] : res[i];
    }
}

// 2 WORD - Step 3:
// 2 Word + popc - scatter
__global__
void reduceChunkBaseline64(uint64_t* real_quote_GPU, uint64_t* prediction_GPU, int total_padded_64){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for(long i = index; i < total_padded_64; i+=stride){
        prediction_GPU[i] = (uint64_t) __popcll(real_quote_GPU[i]);
    }
}


// CUDA kernel where prefix_xor is called
__global__
void inStringFinderBaseline64(uint64_t* real_quote_GPU, uint64_t* prefix_sum_ones, uint64_t* res, int total_padded_64){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < total_padded_64; i += stride){
        // Check if the prefix sum is odd (overflow)
        bool overflow = prefix_sum_ones[i] & 1;

        // Apply prefix_xor64 to real_quote_GPU
        res[i] = prefix_xor64(real_quote_GPU[i]);

        // Update res[i] based on the overflow
        res[i] = overflow ? ~res[i] : res[i];
    }
}

__global__
void findOutUsefulString(uint32_t* op_GPU, uint32_t* newLine_GPU, uint32_t* inString_GPU, uint64_t size, int total_padded_32, int WORDS){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // find all useful character such as { } [ ] : , \n out of sring.
    for(long i = index; i< total_padded_32; i+=stride){
        int start = i*WORDS;
        #pragma unroll
        for(int k=start; k<size && k<start+WORDS; k++){
            uint32_t op = op_GPU[k];                    // { } [ ] : ,
            uint32_t newLine = newLine_GPU[k];    // \n
            uint32_t in_string = inString_GPU[k];

            uint32_t usefulCharacter = op | newLine;
            inString_GPU[k] = ~in_string & usefulCharacter;
        }
    }
}

__global__
void findOutUsefulStringMerge(uint32_t* op_GPU, uint32_t* open_close_GPU, uint32_t* inString_GPU, uint64_t size, int total_padded_32, int WORDS, uint32_t* total_bits){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // find all useful character such as { } [ ] : , \n out of sring.
    for(long i = index; i< total_padded_32; i+=stride){
        int start = i*WORDS;
        #pragma unroll
        for(int k=start; k<size && k<start+WORDS; k++){
            uint32_t all_structural = op_GPU[k];                    // { } [ ] : ,
            uint32_t open_close = open_close_GPU[k];    // \n
            uint32_t in_string = inString_GPU[k];

            inString_GPU[k] = ~in_string & all_structural; // all structural that are out string
            open_close_GPU[k] = ~in_string & open_close; // all open close that are out string

            total_bits[k] = (uint32_t) __popc(inString_GPU[k]);  // total_bits is total_one, we will rename it
            op_GPU[k] = (uint32_t) __popc(open_close_GPU[k]);  // total_bits of open_close is total_one, we will rename it, we put it in op_GPU to prevenet new allocation

        }
    }
}

__global__
void count_set_bits(uint32_t* input, uint32_t* total_bits, int size, uint32_t total_padded_32, int WORDS){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(uint32_t i = index; i< total_padded_32; i+=stride){
        //UPdate
        int start = i*WORDS;
        #pragma unroll
        for(int k=start; k<size && k<start+WORDS; k++){
            total_bits[k] = (uint32_t) __popc(input[k]);
        }
    }
}

    // removeCopy<<<numBlock, BLOCKSIZE>>>(set_bit_count,                      // prefix sum set bits until each word of structural
    //                                     set_bit_count_open_close,           // prefix sum set bits until each word of open close
    //                                     structural_bitmap,                  // structural bitmap out string
    //                                     open_close_GPU,                     // open close bitmap out string
    //                                     block_GPU,                          // real json block
    //                                     out_string_8_GPU,                   // structural byte
    //                                     out_string_8_index_GPU,             // structural real index in real json file
    //                                     out_string_open_close_8_GPU,        // open_close byte
    //                                     out_string_open_close_8_index_GPU,  // structural index for each open close (not real json file)
    //                                     size, 
    //                                     last_index_tokens,                  // structural size
    //                                     last_index_tokens_open_close,       // open close size
    //                                     total_padded_32);

__global__
void removeCopy( uint32_t* set_bit_count,
                 uint32_t* set_bit_count_open_close,
                 uint32_t* out_string, 
                 uint32_t* open_close_GPU, 
                 uint8_t* block_GPU, 
                //  uint8_t* out_string_8_GPU, 
                 uint32_t* out_string_8_index_GPU, 
                 uint8_t* out_string_open_close_8_GPU, 
                 uint32_t* out_string_open_close_8_index_GPU, 
                 uint32_t size, 
                 uint32_t size_structural, 
                 uint32_t size_open_close, 
                 uint32_t total_padded_32,
                 uint64_t lastStructuralIndex,
                 uint64_t lastChunkIndex){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // each thread working on 32 bit, and each bit has 32 bits.
    // each bit is a single character
    // we want to convert it from bit to char
    for(uint32_t i = index; i< total_padded_32; i+=stride){
        uint32_t local_out_string = out_string[i];
        if (local_out_string == 0) continue; 

        uint32_t local_out_string_open_close = open_close_GPU[i];


        uint32_t total_before = i > 0 ? set_bit_count[i-1] : 0;
        uint32_t total_before_open_close = i > 0 ? set_bit_count_open_close[i-1] : 0;

        // uint32_t current_total = 0;
        // uint32_t current_total_open_close = 0;
        
        uint32_t current_total = 0;
        uint32_t current_total_open_close = 0;
        
        uint32_t k = i*32; 

        // Calculate first and last set bit positions
        uint8_t first_set_bit_pos = __ffs(local_out_string) - 1; // Convert to 0-based index
        uint8_t last_set_bit_pos = 32 - __clz(local_out_string) - 1; // Convert to 0-based index

        for (int j = first_set_bit_pos; j <= last_set_bit_pos && k + j < size; j++) {
            uint32_t adjusted_index = total_before + current_total;
            uint32_t adjusted_index_open_close = total_before_open_close + current_total_open_close;

            uint8_t current_bit = (local_out_string >> j) & 1;      
            if (current_bit == 1){
                uint8_t currentChar = block_GPU[k + j];
                // uint8_t replacementChar = (currentChar == 0x0A) ? ',' : currentChar; // check if its new line put comma instead, Check for newline character (0x0A) and set a comma in out_string_8_GPU
                // out_string_8_GPU[adjusted_index] = replacementChar; // Set the value in out_string_8_GPU
                out_string_8_index_GPU[adjusted_index] = k + j + 1 + lastChunkIndex;

                uint8_t current_bit_open_close = (local_out_string_open_close >> j) & 1;      
                if(current_bit_open_close == 1){
                    out_string_open_close_8_GPU[adjusted_index_open_close] = currentChar; // Set the value in out_string_8_GPU
                    out_string_open_close_8_index_GPU[adjusted_index_open_close] = adjusted_index;
                }
                current_total_open_close += current_bit_open_close;       // prefix_sum inside count_set_bits_open_close

            }
            current_total += current_bit;       // prefix_sum inside count_set_bits

        }  
    }
}

inline uint8_t * Tokenize(  uint8_t* block_GPU, 
                            uint64_t size, 
                            int &ret_size, 
                            uint32_t  &last_index_tokens, 
                            uint32_t  &last_index_tokens_open_close, 
                            uint32_t* &in_string_out_index_d,
                            // uint8_t*  &open_close_d,
                            uint32_t* &open_close_index_d,
                            uint64_t lastStructuralIndex,               // last structural index from previous chunk
                            uint64_t lastChunkIndex                     // last real json index from previous chunk
                            ){
    int total_padded_32 = (size+31)/32 ; // size be byte eshe totall padded be bit eshe
    uint8_t*  open_close_d;
    // +31 vase ine k 0 be ma nade o min 1 bde bema
    // va msln size=6 bashe --> 
    //int numBlockBySize = (size + BLOCKSIZE - 1) / BLOCKSIZE;

    // ____________________Initialize________________________
    // uint32_t* general_ptr;
    // hipMallocAsync(&general_ptr, total_padded_32*sizeof(uint32_t)*ROW5, 0);
    // uint32_t* quote_GPU = general_ptr;
    // uint32_t* backslashes_GPU = general_ptr+total_padded_32;
    // uint32_t* newLine_GPU = general_ptr+total_padded_32*ROW2;
    // uint32_t* op_GPU = general_ptr+total_padded_32*ROW3;

    // int WORDS = 2;
    // int total_padded_32B = (size+7)/8;
    // int total_padded_8B = (total_padded_32+1)/2;
    // int total_padded_32_div_32 = (total_padded_32+31)/32;
    // int smallNumBlock = (total_padded_32_div_32 + BLOCKSIZE - 1) / BLOCKSIZE;
    // int numBlock_8B = (total_padded_8B+BLOCKSIZE-1) / BLOCKSIZE;
    // int numBlock_32B    = (total_padded_32B+BLOCKSIZE-1) / BLOCKSIZE;

    // 32 * 8 --> 32

    uint32_t* general_ptr;
    hipMallocAsync(&general_ptr, total_padded_32 * sizeof(uint32_t) * ROW5, 0);
    for (int i = 0; i < ROW5; ++i) {
        uint32_t* row_end_ptr = general_ptr + i * total_padded_32 + (total_padded_32 - 1);
        hipMemsetAsync(row_end_ptr, 0, sizeof(uint32_t), 0);
    }
    hipStreamSynchronize(0);

    uint32_t* quote_GPU         = general_ptr;
    uint32_t* backslashes_GPU   = general_ptr + total_padded_32;
    uint32_t* open_close_GPU    = general_ptr + total_padded_32 * ROW2;
    uint32_t* op_GPU            = general_ptr + total_padded_32 * ROW3;

    int WORDS = 2;

    int total_padded_8B         = (total_padded_32 + 1) / 2;
    int total_padded_16B        = (total_padded_32 + 3) / 4;
    int total_padded_32_div_8   = (total_padded_32 + 7) / 8;
    int total_padded_32_div_32  = (total_padded_32 + 31) / 32;

    int total_padded_8 = (size + 7) / 8;
    int total_padded_32B = (size + 7) / 8;
    // int total_padded_32 = (size + 31) / 32; // most used
    int total_padded_64 = (size + 63) / 64;

    int smallNumBlock   = (total_padded_32_div_32 + BLOCKSIZE - 1) / BLOCKSIZE;
    int smallNumBlock_8 = (total_padded_32_div_8 + BLOCKSIZE - 1) / BLOCKSIZE;

    int numBlock        = (total_padded_32 + BLOCKSIZE - 1) / BLOCKSIZE;
    int numBlock_8      = (total_padded_8 + BLOCKSIZE - 1) / BLOCKSIZE;
    int numBlock_8B     = (total_padded_8B + BLOCKSIZE - 1) / BLOCKSIZE;
    int numBlock_16B    = (total_padded_16B + BLOCKSIZE - 1) / BLOCKSIZE;
    int numBlock_32B    = (total_padded_32B + BLOCKSIZE - 1) / BLOCKSIZE;
    int numBlock_64     = (total_padded_64 + BLOCKSIZE - 1) / BLOCKSIZE;


    // Prepare
    //t hipEvent_t start, stop;
    //t hipEventCreate(&start);
    //t hipEventCreate(&stop);
    // Start record
    //t hipEventRecord(start, 0);

    // auto start = chrono::high_resolution_clock::now();
    // // __________________Create_Bit-Map_Character___________________
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    // Step 1
    bitMapCreatorSimd<<<numBlock_8, BLOCKSIZE>>>( (uint32_t*) block_GPU, (uint8_t*) backslashes_GPU, (uint8_t*) quote_GPU, (uint8_t*) op_GPU, (uint8_t*) open_close_GPU, size, total_padded_8);
    hipStreamSynchronize(0);



    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Step 1 Time: " << milliseconds << " ms" << std::endl;

    // cout << "back_slash: \n";
    // print_d32(backslashes_GPU,total_padded_32,ROW1); 
    // cout << "dquote: \n";
    // print_d32(quote_GPU,total_padded_32,ROW1); 
    // cout << "op: \n";
    // print_d32(op_GPU,total_padded_32,ROW1); 
    // cout << "op: \n";
    // print_d32(open_close_GPU,total_padded_32,ROW1); 

    // __________________Find_Escaped_Character_____________________
    // Step 2
    uint32_t* real_quote_GPU = general_ptr + total_padded_32 * ROW4;
    // hipEventRecord(start, 0);
    
    
    // findEscapedQuote<<<numBlock_8B, BLOCKSIZE>>>(backslashes_GPU, quote_GPU, real_quote_GPU, total_padded_32, total_padded_8B, WORDS);
    findEscapedQuoteMerge_NEW<<<numBlock_8B, BLOCKSIZE>>>(backslashes_GPU, quote_GPU, real_quote_GPU, total_padded_32, total_padded_8B, WORDS);
    hipStreamSynchronize(0);

    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Step 2 Time: " << milliseconds << " ms" << std::endl;
 
    // cout << "Time taken by program is Token Escaped Quote [step-2] : " << fixed << time_taken2 << setprecision(9);
    // cout << " sec" << endl;
    // print_d32(real_quote_GPU,total_padded_32,ROW1);
    // printf("findEscapedQuote Works Well!\n");


    // cout << "real quote: \n";
    // print_d32(real_quote_GPU,total_padded_32,ROW1); 
    // cout << "popc quote: \n";
    // print_d32(quote_GPU,total_padded_32,ROW1); 

    // Step 3a
    // __________________________REDUCE______________________________
     // Step 3a
    uint32_t* total_one_GPU = general_ptr;
    // uint32_t* total_one_32_GPU;
    // hipMallocAsync(&total_one_32_GPU, (total_padded_32_div_32) * sizeof(uint32_t), 0);
    // hipMallocAsync(&total_one_32_GPU, (total_padded_32_div_8) * sizeof(uint32_t), 0);
    // hipEventRecord(start, 0);
    // reduceChunkBaseline<<<numBlock, BLOCKSIZE>>>(real_quote_GPU, total_one_GPU, total_padded_32);
    // reduceChunkBaseline64<<<numBlock_64, BLOCKSIZE>>>((uint64_t*) real_quote_GPU, (uint64_t*) total_one_GPU, total_padded_64);
    // reduceChunkBaseline4Words<<<numBlock_16B, BLOCKSIZE>>>(real_quote_GPU, total_one_GPU, total_padded_16B, size);
    // hipStreamSynchronize(0);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Step 3a Time: " << milliseconds << " ms" << std::endl;
    // cout << "popc quote: \n";
    // print_d32(total_one_GPU,total_padded_32,ROW1); 
    
    // Step 3b
    // hipEventRecord(start, 0);
    // thrust::exclusive_scan(thrust::cuda::par, (uint64_t*) total_one_GPU, ( (uint64_t*) total_one_GPU ) + (total_padded_64), (uint64_t*) total_one_GPU);
    thrust::exclusive_scan(thrust::cuda::par, total_one_GPU, total_one_GPU + (total_padded_32), total_one_GPU);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Step 3b Time: " << milliseconds << " ms" << std::endl;


    // cout << "scan popc quote: \n";
    // print_d32(total_one_GPU,total_padded_32,ROW1); 

    // Step 3d
    uint32_t* inString_GPU = general_ptr;

    // hipEventRecord(start, 0);
    inStringFinderBaseline<<<numBlock, BLOCKSIZE>>>(real_quote_GPU, total_one_GPU, inString_GPU, total_padded_32);
    // inStringFinderBaseline64<<<numBlock_64, BLOCKSIZE>>>((uint64_t*) real_quote_GPU, (uint64_t*) total_one_GPU, (uint64_t*) inString_GPU, total_padded_64);
    hipStreamSynchronize(0);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Step 3d Time: " << milliseconds << " ms" << std::endl;

    // cout << "open close: \n";
    // print_d32(open_close_GPU,total_padded_32,ROW1); 


    // cout << "in string popc quote: \n";
    // print_d32(inString_GPU,total_padded_32,ROW1); 
    // exit(0);
    // Step 4
    // hipEventRecord(start, 0);
    // findOutUsefulString<<<numBlock_8B, BLOCKSIZE>>>(op_GPU, newLine_GPU, inString_GPU, total_padded_32, total_padded_8B, WORDS);
    // hipStreamSynchronize(0);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Step 4 Time: " << milliseconds << " ms" << std::endl;

    // Step 4 merge with 5a
    uint32_t* set_bit_count = general_ptr + total_padded_32;
    // hipEventRecord(start, 0);
    findOutUsefulStringMerge<<<numBlock_8B, BLOCKSIZE>>>(op_GPU, open_close_GPU, inString_GPU, total_padded_32, total_padded_8B, WORDS, set_bit_count);
    hipStreamSynchronize(0);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Step 4 Time: " << milliseconds << " ms" << std::endl;

    uint32_t* set_bit_count_open_close = op_GPU; // lets rename it for easy understanding
    uint32_t* structural_bitmap = inString_GPU;

    // cout << "out string quote: \n";
    // print_d32(inString_GPU,total_padded_32,ROW1); 
    // cout << "out string oc: \n";
    // print_d32(open_close_GPU,total_padded_32,ROW1); 
    // cout << "set bit counts quote: \n";
    // print_d32(set_bit_count,total_padded_32,ROW1); 
    // cout << "set bit counts open close: \n";
    // print_d32(set_bit_count_open_close,total_padded_32,ROW1); 
    // exit(0);

 

 
    // cout << "Time taken by program is Token [step-4] : " << fixed << time_taken7 << setprecision(9);
    // cout << " sec" << endl;
    // print_d(inString_GPU,total_padded_32,ROW1); 
    // printf("findOutUsefulString Works Well!\n");
    
    // ______________Final_Step_Write_____________________
    // step 5a:
    // uint32_t* set_bit_count = general_ptr + total_padded_32;
    // hipEventRecord(start, 0);
    // count_set_bits<<<numBlock_8B, BLOCKSIZE>>>(structural_bitmap, set_bit_count, total_padded_32, total_padded_8B, WORDS);
    // hipStreamSynchronize(0);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Step 5a Time: " << milliseconds << " ms" << std::endl;

    // Step 5b
    // hipEventRecord(start, 0);
    thrust::inclusive_scan(thrust::cuda::par, set_bit_count, set_bit_count + total_padded_32, set_bit_count);
    hipMemcpyAsync(&last_index_tokens, set_bit_count + total_padded_32 - 1, sizeof(uint32_t), hipMemcpyDeviceToHost);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Step 5b Time: " << milliseconds << " ms" << std::endl;

    // Step 5c
    // hipEventRecord(start, 0);
    thrust::inclusive_scan(thrust::cuda::par, set_bit_count_open_close, set_bit_count_open_close + total_padded_32, set_bit_count_open_close);
    hipMemcpyAsync(&last_index_tokens_open_close, set_bit_count_open_close + total_padded_32 - 1, sizeof(uint32_t), hipMemcpyDeviceToHost);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Step 5c Time: " << milliseconds << " ms" << std::endl;


    // Step 5d
    // last_index_tokens += 3;
    // last_index_tokens_open_close += 2;

    int reminder = last_index_tokens % 4;    
    int padding = (4-reminder) & 3; 
    // It will always return a number between 0 and 3, 
    // which represents the number of padding bytes needed to align the size to the next multiple of 4.
    // uint64_t last_index_tokens_padded = (last_index_tokens + padding)/4;



    // uint8_t* out_string_8_GPU;
    uint32_t* out_string_8_index_GPU; // it's going to store real index.
    // hipMallocAsync(&out_string_8_GPU, (last_index_tokens + padding) * sizeof(uint8_t),0);

    hipMallocAsync(&out_string_8_index_GPU, last_index_tokens * sizeof(uint32_t) * ROW2,0); // Row 1 for structural index, Row 2 for ending pos which will calculated in parsr

    int reminder2 = last_index_tokens_open_close % 4;    
    int padding2 = (4-reminder2) & 3; 
    // It will always return a number between 0 and 3, 
    // which represents the number of padding bytes needed to align the size to the next multiple of 4.
    // uint64_t last_index_tokens_open_close_padded = (last_index_tokens_open_close + padding2)/4;



    uint8_t* out_string_open_close_8_GPU;
    uint32_t* out_string_open_close_8_index_GPU; // it's going to store structural index, not real index
    hipMallocAsync(&out_string_open_close_8_GPU, (last_index_tokens_open_close + padding2)  * sizeof(uint8_t),0);
    hipMallocAsync(&out_string_open_close_8_index_GPU, last_index_tokens_open_close * sizeof(uint32_t),0);

    // cout << "res size before remove copy: " << last_index_tokens_open_close << "\n";

    // hipEventRecord(start, 0);
    removeCopy<<<numBlock, BLOCKSIZE>>>(set_bit_count,                      // prefix sum set bits until each word of structural
                                        set_bit_count_open_close,           // prefix sum set bits until each word of open close
                                        structural_bitmap,                  // structural bitmap out string
                                        open_close_GPU,                     // open close bitmap out string
                                        block_GPU,                          // real json block
                                        // out_string_8_GPU,                   // structural byte
                                        out_string_8_index_GPU,             // structural real index in real json file
                                        out_string_open_close_8_GPU,        // open_close byte
                                        out_string_open_close_8_index_GPU,  // structural index for each open close (not real json file)
                                        size, 
                                        last_index_tokens,                  // structural size
                                        last_index_tokens_open_close,       // open close size
                                        total_padded_32,
                                        lastStructuralIndex,                // last structural index from previous chunk
                                        lastChunkIndex);                    // last real json index from previous chunk
    hipStreamSynchronize(0);
    
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Step 5d Time: " << milliseconds << " ms" << std::endl;

    // hipMemcpyAsync(&last_index_tokens, set_bit_count+total_padded_32-1, sizeof(uint32_t), hipMemcpyDeviceToHost);
    // last_index_tokens += 3;
    hipFreeAsync(general_ptr,0);


    in_string_out_index_d = out_string_8_index_GPU;
    // uint8_t* in_string_out_d;
    // in_string_out_d = out_string_8_GPU;
    ret_size = last_index_tokens; // latest index toye vagheait data

    open_close_d = out_string_open_close_8_GPU;
    open_close_index_d = out_string_open_close_8_index_GPU;

    // cout << "res size after remove copy: " << last_index_tokens_open_close << "\n";
    // print8_d<uint8_t>(in_string_out_d,ret_size,ROW1); 
    // exit(0);

    // cout << "res size: " << last_index_tokens_open_close << "\n";
    // print8_d<uint8_t>(open_close_d,last_index_tokens_open_close,ROW1); 
    // exit(0);

    // hipEventRecord(start, 0);
    // cout << "index-after sort by key" << endl;
    // printUInt32ArrayFromGPU( open_close_index_d, last_index_tokens_open_close);

    // printf("removeCopy Works Well!\n");
    // cout << length << endl;
    // printInt32ArrayFromGPU(Row3Start, length);

    // exit(0);
    // hipEventDestroy(start);
    // hipEventDestroy(stop);

    return open_close_d;
}

__global__
void depth_init_MathAPI(uint32_t* open_close_GPU, uint32_t* oc_1, int oc_cnt_32, int oc_cnt){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int32_t i = index; i < oc_cnt_32 && i < oc_cnt ; i+=stride){
        uint32_t idx = i*4;
        uint32_t current_4_bytes = open_close_GPU[i];

        uint32_t isOpen = (__vcmpeq4(current_4_bytes, 0x5B5B5B5B) | __vcmpeq4(current_4_bytes, 0x7B7B7B7B) ) & 0x01010101; // 01
        uint32_t isClose = (__vcmpeq4(current_4_bytes, 0x5D5D5D5D) |  __vcmpeq4(current_4_bytes, 0x7D7D7D7D) );            // FF

        oc_1[i] = (isOpen |  isClose); 
    }
}

__global__
void validate_expand_MathAPI_new(char* structural_GPU, uint32_t* index_arr, uint32_t* endIdx, int oc_cnt_32, int oc_cnt, bool* error){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; 
    // [ ] { } [ ] { }
    // 0 9 1 8 4 5 6 7
    __shared__ uint32_t shared_error;
    if(threadIdx.x == 0) shared_error = 0;
    __syncthreads();

    for(int32_t i = index; i < oc_cnt_32; i+=stride){
        int k = i*4; 
        int currentIndex = index_arr[k];

        if( i == oc_cnt_32 - 1){
            // printf("outside error: %c\n", structural_GPU[currentIndex]);
            if(k+1 >= oc_cnt){
                // printf("here error1: %c\n", structural_GPU[currentIndex]);
                atomicOr(&shared_error, 1); 
            }else if(k+2 >= oc_cnt){
                // printf("here error2: %c\n", structural_GPU[currentIndex]);
                int nextIndex = index_arr[k+1];
                uint32_t two_chars = structural_GPU[currentIndex] | structural_GPU[nextIndex] << 8;
                // [ ]
                // { }
                uint32_t error_local = (__vcmpeq2(two_chars, 0x5D5B) | __vcmpeq2(two_chars, 0x7D7B));
                atomicOr(&shared_error, ~error_local & 0x1); 
                endIdx[currentIndex] = nextIndex;
            }else if(k+3 >= oc_cnt){
                // printf("here error3: %c\n", structural_GPU[currentIndex]);
                atomicOr(&shared_error, 1); 
            }else{
                // printf("here error4: %c\n", structural_GPU[currentIndex]);
                int nextIndex = index_arr[k+1];
                int currentIndex_2 = index_arr[k+2];
                int nextIndex_2 = index_arr[k+3];
            
                uint32_t four_chars = structural_GPU[currentIndex] | structural_GPU[nextIndex] << 8 | structural_GPU[currentIndex_2] << 16 | structural_GPU[nextIndex_2] << 24;
                uint32_t shifted_four_chars = four_chars << 8;
                uint32_t xor_chars =  (four_chars ^ shifted_four_chars) & 0xFF00FF00;        
                uint32_t error_local = __vcmpeq4(xor_chars, 0x06000600);
                atomicOr(&shared_error, ~error_local > 0); 

                endIdx[currentIndex] = nextIndex;
                endIdx[currentIndex_2] = nextIndex_2;
            }   
        }else{    
            // printf("here error4: %c\n", structural_GPU[currentIndex]);
            int nextIndex = index_arr[k+1];

            int currentIndex_2 = index_arr[k+2];
            int nextIndex_2 = index_arr[k+3];
            
            // 5b xor 5d = 06
            // 7b xor 7d = 06
            uint32_t four_chars = structural_GPU[currentIndex] | structural_GPU[nextIndex] << 8 | structural_GPU[currentIndex_2] << 16 | structural_GPU[nextIndex_2] << 24;
            uint32_t shifted_four_chars = four_chars << 8;
            uint32_t xor_chars =  (four_chars ^ shifted_four_chars) & 0xFF00FF00;        
            uint32_t error_local = __vcmpeq4(xor_chars, 0x06000600) & 0xFFFFFFFF;

            atomicOr(&shared_error, ~error_local > 0); 

            endIdx[currentIndex] = nextIndex;
            endIdx[currentIndex_2] = nextIndex_2;
        }
        __syncthreads();
        if (threadIdx.x == 0 && shared_error) *error = true;
    }

}

__global__
void validate_expand_MathAPI_new2(char* pair_oc, uint32_t* index_arr, uint32_t* endIdx, int oc_cnt_32, int oc_cnt, bool* error, uint64_t lastStructuralIndex){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; 
    // [ ] { } [ ] { }
    // 0 9 1 8 4 5 6 7
    __shared__ uint32_t shared_error;
    if(threadIdx.x == 0) shared_error = 0;
    __syncthreads();

    for(int32_t i = index; i < oc_cnt_32; i+=stride){
        int k = i*4; 
        // int currentIndex = index_arr[k];

        if( i == oc_cnt_32 - 1){
            // printf("outside error: %c\n", structural_GPU[currentIndex]);
            if(k+1 >= oc_cnt){
                // printf("here error1: %c\n", structural_GPU[currentIndex]);
                shared_error |= 1; 
            }else if(k+2 >= oc_cnt){
                // printf("here error1: %c\n", pair_oc[k]);
                // printf("here error2: %c\n", pair_oc[k+1]);
                uint32_t two_chars = pair_oc[k] | pair_oc[k+1] << 8;
                // printf("32: %x\n", two_chars);
                uint32_t shifted_two_char = two_chars << 8; 
                // printf("shifted_two_char 32: %x\n", shifted_two_char);
                uint32_t xor_chars =  (two_chars ^ shifted_two_char) & 0x0000FF00;        
                // printf("xor: %x\n", xor_chars);
                uint32_t error_local = __vcmpeq4(xor_chars, 0x00000600);
                // printf("err: %x\n", error_local);
                shared_error |= (~error_local) > 0; 

                endIdx[index_arr[k]] = index_arr[k+1] + lastStructuralIndex + 1;

            }else if(k+3 >= oc_cnt){
                // printf("here error3: %c\n", structural_GPU[currentIndex]);
                // atomicOr(&shared_error, 1); 
                shared_error |= 1; 
            }else{
                // printf("here error4: %c\n", structural_GPU[currentIndex]);
                // int nextIndex = index_arr[k+1];
                // int currentIndex_2 = index_arr[k+2];
                // int nextIndex_2 = index_arr[k+3];
            
                uint32_t four_chars = pair_oc[k] | pair_oc[k+1] << 8 | pair_oc[k+2] << 16 | pair_oc[k+3] << 24;
                uint32_t shifted_four_chars = four_chars << 8;
                uint32_t xor_chars =  (four_chars ^ shifted_four_chars) & 0xFF00FF00;        
                uint32_t error_local = __vcmpeq4(xor_chars, 0x06000600) & 0xFFFFFFFF;
                shared_error |= (~error_local) > 0; 

                endIdx[index_arr[k]] = index_arr[k+1] + lastStructuralIndex + 1;
                endIdx[index_arr[k+2]] = index_arr[k+3] + lastStructuralIndex + 1;
            }   
        }else{    
            // printf("here error4: %c\n", structural_GPU[currentIndex]);
            // int nextIndex = index_arr[k+1];

            // int currentIndex_2 = index_arr[k+2];
            // int nextIndex_2 = index_arr[k+3];
            
            // 5b xor 5d = 06
            // 7b xor 7d = 06
            
            uint32_t order_err = pair_oc[k] > pair_oc[k+1] | pair_oc[k+2] > pair_oc[k+3];
            uint32_t four_chars = pair_oc[k] | pair_oc[k+1] << 8 | pair_oc[k+2] << 16 | pair_oc[k+3] << 24;
            uint32_t shifted_four_chars = four_chars << 8;
            uint32_t xor_chars =  (four_chars ^ shifted_four_chars) & 0xFF00FF00;        
            uint32_t error_local = __vcmpeq4(xor_chars, 0x06000600) & 0xFFFFFFFF;

            shared_error |= (~error_local | order_err) > 0; 

            // atomicOr(&shared_error, (~error_local | order_err) > 0); 

            endIdx[index_arr[k]] = index_arr[k+1] + lastStructuralIndex + 1;
            endIdx[index_arr[k+2]] = index_arr[k+3] + lastStructuralIndex + 1;
        }
        __syncthreads();
        if (threadIdx.x == 0 && shared_error) *error = true;
    }

}


void printByteByByte(int32_t* data, int length) {
    for (int i = 0; i < length; ++i) {
        unsigned char* bytePointer = (unsigned char*)&data[i];
        for (int j = 0; j < sizeof(int32_t); ++j) {
            printf("%02x ", bytePointer[j]);
        }
        printf("\n");
    }
}

// int32_t* Parser(uint8_t* open_close_GPU, char* structural_GPU, int32_t** open_close_index_d,  int32_t** real_input_index_d, int oc_cnt, int structural_cnt, int & result_size) {
int32_t* Parser(uint8_t* open_close_GPU, int32_t** open_close_index_d,  int32_t** real_input_index_d, int oc_cnt, int structural_cnt, int & result_size, uint64_t lastStructuralIndex) {
    //cout << "PARSING START!\n";
    //char inputTestSample[] = { '{', ':', ',', ':', ',', ':', '[', ',', ',', ',', ']', ',', ':', ',', ':', '}', ','};
    //char* inputTest = inputTestSample;
    //int structural_cnt = sizeof(inputTest)/sizeof(inputTest[0]);

    // int reminder = structural_cnt % 4;    
    // int padding = (4-reminder) & 3; 
    // // It will always return a number between 0 and 3, 
    // // which represents the number of padding bytes needed to align the size to the next multiple of 4.
    // uint64_t structural_cnt_padded = (structural_cnt + padding)/4;



    // int reminder2 = oc_cnt % 4;    
    // int padding2 = (4-reminder2) & 3; 
    // // It will always return a number between 0 and 3, 
    // // which represents the number of padding bytes needed to align the size to the next multiple of 4.
    // uint64_t oc_cnt_padded = (oc_cnt + padding2)/4;


    uint32_t* oc_idx = reinterpret_cast<uint32_t*>(*open_close_index_d);
    uint32_t* parsed_oc = reinterpret_cast<uint32_t*>(*real_input_index_d); // contains two rows--> 1. structural     2. pair_pos 

    // cout << "index-after sort by key" << endl;
    // printUInt32ArrayFromGPU( oc_idx, oc_cnt);

    // stat
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    // cout << oc_cnt << endl;
    // printCharArrayFromGPU(open_close_GPU, oc_cnt);

    // print8_d<uint8_t>(open_close_GPU,oc_cnt,ROW1); 
 
    
    // _______________STEP_1__(a)_________________    
    // int numBlock = (structural_cnt + BLOCKSIZE - 1) / BLOCKSIZE;
    // int numBlock_open_close = (oc_cnt + BLOCKSIZE - 1) / BLOCKSIZE;

    // int WORDS = 4;
    // int structural_cnt_32 = (structural_cnt + WORDS - 1) / WORDS;                   // for times that we are working on 4 bytes instead of 1 bytes in a thread
    // int numBlock_32 = (structural_cnt_32 + BLOCKSIZE - 1) / BLOCKSIZE;

    // int oc_cnt_32 = (oc_cnt + WORDS - 1) / WORDS;
    // int numBlock_open_close_32 = (oc_cnt_32 + BLOCKSIZE - 1) / BLOCKSIZE;



    int numBlock = (structural_cnt + BLOCKSIZE - 1) / BLOCKSIZE;
    int numBlock_open_close = (oc_cnt + BLOCKSIZE - 1) / BLOCKSIZE;

    int WORDS = 4;
    int structural_cnt_32 = (structural_cnt + WORDS - 1) / WORDS;                   // for times that we are working on 4 bytes instead of 1 bytes in a thread
    int numBlock_32 = (structural_cnt_32 + BLOCKSIZE - 1) / BLOCKSIZE;

    int oc_cnt_32 = (oc_cnt + WORDS - 1) / WORDS;
    int numBlock_open_close_32 = (oc_cnt_32 + BLOCKSIZE - 1) / BLOCKSIZE;


    // int oc_cnt_32 = (oc_cnt + WORDS - 1) / WORDS;
    // int numBlock_open_close_32 = (oc_cnt_32 + BLOCKSIZE - 1) / BLOCKSIZE;

    int32_t* res; // temporary result that will use in following

    // hipEventRecord(start, 0);


    uint32_t* oc_1; // output 
    hipMallocAsync(&oc_1, oc_cnt_32*sizeof(uint32_t), 0); 

    // uint32_t* parsed_oc = ; // ROW0 --> idx   ROW1 --> end (output)
    // hipMallocAsync(&parsed_oc, structural_cnt*ROW2*sizeof(uint32_t), 0); 
    
    depth_init_MathAPI<<<numBlock_open_close_32, BLOCKSIZE>>>( (uint32_t*) open_close_GPU, oc_1, oc_cnt_32, oc_cnt);
    hipStreamSynchronize(0);

    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);

    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Parser Time taken by [step-1-a] : %f ms\n", milliseconds);


    // cout << "open close" << endl;
    // printUInt8ArrayFromGPU( (uint8_t*) Row2Start, structural_cnt);

    // cout << "depth" << endl;
    // printUInt8ArrayFromGPU( (uint8_t*) arr, structural_cnt);

    // cout << "index" << endl;
    // printUInt32ArrayFromGPU( idx, structural_cnt);

    // print_d(arr,structural_cnt,ROW3);
    // print8_d<uint8_t>(arr,structural_cnt,ROW3); 

    // exit(0);

    // hipEventRecord(start, 0);


    uint32_t* depth = oc_1; // output 
    // // _______________STEP_1__(b)_________________
    thrust::inclusive_scan(thrust::cuda::par,  (uint8_t*) depth,  ((uint8_t*) depth) + oc_cnt,  (uint8_t*) depth); // on depth

    // cout << "count = " << oc_cnt <<endl;
    // cout << "depth-scan" << endl;
    // printUInt8ArrayFromGPU( (uint8_t*) depth, oc_cnt);

    // exit(0);

    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Parser Time taken by [step-1-b] : %f ms\n", milliseconds);

    // hipEventRecord(start, 0);

    // // _______________STEP_2__(a)_________________
    // hipMemcpyAsync(Row3Start, arr, sizeof(uint32_t)*structural_cnt_32, hipMemcpyDeviceToDevice, 0); 
    thrust::transform_if(thrust::cuda::par, (uint8_t*) depth, ((uint8_t*) depth) + oc_cnt, open_close_GPU, (uint8_t*) depth, decrease(), is_opening());

    // cout << "count = " << oc_cnt <<endl;
    // cout << "depth-scan" << endl;
    // printUInt8ArrayFromGPU( (uint8_t*) depth, oc_cnt);


    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Parser Time taken by [step-2-a] : %f ms\n", milliseconds);


    // // _______________STEP_3__(b)_________________
    // thrust::stable_sort_by_key(thrust::cuda::par, (uint8_t*) depth,  ((uint8_t*) depth) + oc_cnt, oc_idx);

    // Use zip iterator to combine oc_idx and open_close_GPU
    auto zipped_begin = thrust::make_zip_iterator(thrust::make_tuple(oc_idx, open_close_GPU));
    // auto zipped_end = thrust::make_zip_iterator(thrust::make_tuple(oc_idx + oc_cnt, open_close_GPU + oc_cnt));

    // Sorting based on depth using a single stable_sort_by_key
    thrust::stable_sort_by_key(thrust::cuda::par, (uint8_t*)depth, ((uint8_t*)depth) + oc_cnt, zipped_begin);

    char* pair_oc = (char *) open_close_GPU;
    uint32_t* pair_idx = oc_idx;
    // cout << "depth-scan" << endl;
    // printUInt8ArrayFromGPU( (uint8_t*) depth, oc_cnt);
    // cout << "index-after sort by key" << endl;
    // printUInt32ArrayFromGPU( oc_idx, oc_cnt);
    // cout << "index-after sort by key" << endl;
    // printUInt32ArrayFromGPU( open_close_GPU, oc_cnt);
    // print_d(open_close_GPU,oc_cnt,ROW1);
    // print8_d<uint8_t>(open_close_GPU,oc_cnt,ROW1); 



    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Parser Time taken by [step-3-b] : %f ms\n", milliseconds);

    // hipEventRecord(start, 0);

    // _______________STEP_4__(a)_________________
    // int error = 0;

    bool pairError = false;
    bool* pairError_GPU;
    hipMallocAsync(&pairError_GPU, sizeof(bool), 0);                  //  Allocates Memory on the Device and Returns a Pointer to the Allocated Memory.
    hipMemsetAsync(pairError_GPU, 0, sizeof(bool), 0);                //  Initializes a Block of Memory on the Device with a Specified Value
  
    uint32_t* pair_pos = parsed_oc + structural_cnt;
    // validate_expand_MathAPI_new<<<numBlock_open_close_32, BLOCKSIZE>>>(structural_GPU, pair_idx, end_pos, oc_cnt_32, oc_cnt, pairError_GPU); 
    validate_expand_MathAPI_new2<<<numBlock_open_close_32, BLOCKSIZE>>>(pair_oc, pair_idx, pair_pos, oc_cnt_32, oc_cnt, pairError_GPU, lastStructuralIndex); 

    hipStreamSynchronize(0);
    hipMemcpyAsync(&pairError, pairError_GPU, sizeof(bool), hipMemcpyDeviceToHost, 0);

    if(pairError){  // 0 no error, 1 error
        printf("error found!");
        exit(0);
    }




    // hipMemcpyAsync(parsed_oc, *real_input_index_d, structural_cnt*sizeof(uint32_t), hipMemcpyDeviceToDevice,0);
    // hipFreeAsync(*real_input_index_d, 0);
    result_size = structural_cnt;


    // cout << "index-after real json" << endl;
    // printUInt32ArrayFromGPU(parsed_oc, structural_cnt);
    // printUInt32ArrayFromGPU(parsed_oc + structural_cnt, structural_cnt);


    // exit(0);

    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Parser Time taken by [step-4-a] : %f ms\n", milliseconds);

    // hipEventDestroy(start);
    // hipEventDestroy(stop);

    // hipFreeAsync(*open_close_index_d, 0);
    hipFreeAsync(open_close_GPU, 0);
    // hipFreeAsync(structural_GPU, 0);
    hipFreeAsync(depth, 0);

    return (int32_t*) parsed_oc;
    // return NULL;
    //arr(output): ROW 1 depth (not anymore) | ROW1 Real Character Index | ROW2 End Index (for each opening)
}

// block_GPU is block_GPU
inline void *start(void* inputStart){
    // _________________INIT_________________________
    uint8_t* block = ((inputStartStruct *)inputStart)->block;
    uint64_t size = ((inputStartStruct *)inputStart)->size;
    uint64_t lastStructuralIndex = ((inputStartStruct *)inputStart)->lastStructuralIndex;
    uint64_t lastChunkIndex = ((inputStartStruct *)inputStart)->lastChunkIndex;




    uint8_t* block_GPU;        // BLOCKS in GPU
    // uint8_t* tokens_GPU;        // TOKEN RESULT for GPU
    uint8_t* open_close_GPU;
    uint64_t * parse_tree; 

    //printf("block in start:\n %s \n", block);
    //printf("size: %d\n", size);
    int reminder = size%4;    
    int padding = (4-reminder) & 3; 
    // It will always return a number between 0 and 3, which represents the number of padding bytes needed to align the size to the next multiple of 4.
    uint64_t size_32 = (size + padding)/4;



    hipMallocAsync(&block_GPU, (size+padding)*sizeof(uint8_t),0);
    hipMemsetAsync(block_GPU, 0, (size+padding)*sizeof(uint8_t), 0);
    ////////////////Time
    hipEvent_t startHD, stopHD;
    hipEventCreate(&startHD);
    hipEventCreate(&stopHD);
    hipEventRecord(startHD, 0);

    hipMemcpyAsync(block_GPU, block, sizeof(uint8_t)*size, hipMemcpyHostToDevice, 0);

    hipEventRecord(stopHD, 0);
    hipEventSynchronize(stopHD);
    float elapsedTimeHD;
    hipEventElapsedTime(&elapsedTimeHD, startHD, stopHD);

    time_EE.copy_start += elapsedTimeHD;





    // hipEvent_t startEE, stopEE;
    // hipEventCreate(&startEE);
    // hipEventCreate(&stopEE);
    // hipEventRecord(startEE, 0);

    hipEvent_t startValEE, stopValEE;
    hipEventCreate(&startValEE);
    hipEventCreate(&stopValEE);
    hipEventRecord(startValEE, 0);

    // _________________Validation___________________
    bool isValidUTF8 = UTF8Validation(reinterpret_cast<uint32_t *>(block_GPU), size_32);
    hipStreamSynchronize(0);
    //printf("Success before if - isValidUTF8 = %d\n",isValidUTF8);
    if(!isValidUTF8) {
        //printf("not a valid UTF input- isValidUTF8 = %d\n",isValidUTF8); 
        exit(0);
    }

    // hipEventRecord(stopValEE, 0);
    // hipEventSynchronize(stopValEE);
    float elapsedTimeVal;
    // hipEventElapsedTime(&elapsedTimeVal, startValEE, stopValEE);

    time_EE.EE_t_val += elapsedTimeVal;
    time_EE.EE_t += elapsedTimeVal;


    // __________________Tokenizer___________________
    // hipEvent_t startTokEE, stopTokEE;
    // hipEventCreate(&startTokEE);
    // hipEventCreate(&stopTokEE);
    // hipEventRecord(startTokEE, 0);
    

    uint32_t last_index_tokens;
    uint32_t last_index_tokens_open_close;
    int ret_size = 0;
    uint32_t* tokens_index_GPU;
    uint32_t* open_close_index_GPU;
    // tokens_GPU = Tokenize(block_GPU, size, ret_size, last_index_tokens, last_index_tokens_open_close, tokens_index_GPU, open_close_GPU, open_close_index_GPU);
    open_close_GPU = Tokenize(block_GPU, size, ret_size, last_index_tokens, last_index_tokens_open_close, tokens_index_GPU, open_close_index_GPU, lastStructuralIndex, lastChunkIndex);
    // hipStreamSynchronize(0);

    // hipEventRecord(stopTokEE, 0);
    // hipEventSynchronize(stopTokEE);
    // float elapsedTimeTok;
    // hipEventElapsedTime(&elapsedTimeTok, startTokEE, stopTokEE);

    // time_EE.EE_t_tok += elapsedTimeTok;
    // time_EE.EE_t += elapsedTimeTok;



    

    // hipEventRecord(start, 0);
    // cout << "index-after sort by key [after tok]" << endl;
    // printUInt32ArrayFromGPU( open_close_index_GPU, last_index_tokens_open_close);

    int32_t* result_GPU;
    int32_t* result;
    int result_size;

    // hipStreamSynchronize(0);
    // __________________Parsing_____________________
    // hipEvent_t startParseEE, stopParseEE;
    // hipEventCreate(&startParseEE);
    // hipEventCreate(&stopParseEE);
    // hipEventRecord(startParseEE, 0);

    // result_GPU = Parser((char *)tokens_GPU, (int32_t **)(&tokens_index_GPU),  last_index_tokens, result_size);
    // result_GPU = Parser(open_close_GPU, 
    //                     (char *)tokens_GPU, 
    //                     (int32_t **)(&open_close_index_GPU), 
    //                     (int32_t **)(&tokens_index_GPU), 
    //                     last_index_tokens_open_close, 
    //                     last_index_tokens, 
    //                     result_size);
    // ((inputStartStruct *)inputStart)->result_size = result_size;
    result_GPU = Parser(open_close_GPU, 
                        (int32_t **)(&open_close_index_GPU), 
                        (int32_t **)(&tokens_index_GPU), 
                        last_index_tokens_open_close, 
                        last_index_tokens, 
                        result_size,
                        lastStructuralIndex);

    ((inputStartStruct *)inputStart)->result_size = result_size;


    // cout << "result size here: " << result_size <<endl; 
    uint32_t total_tokens = (uint32_t) last_index_tokens;
    // cout << total_tokens << endl;
    uint32_t total_result_size = (uint32_t) result_size*ROW2;

    // hipEventRecord(stopParseEE, 0);
    // hipEventSynchronize(stopParseEE);
    // float elapsedTimeParse;
    // hipEventElapsedTime(&elapsedTimeParse, startParseEE, stopParseEE);

    // time_EE.EE_t_pars += elapsedTimeParse;
    // time_EE.EE_t += elapsedTimeParse;


    hipFreeAsync(block_GPU,0); 

    return (void *)result_GPU;
    
}

inline int32_t *readFileLine(char *file,int n, resultStructGJSON* resultStruct){
    // _________________INIT_________________________
    unsigned long  bytesread;
    static uint8_t*  buf;    // gpu
    static int32_t* res_buf; // cpu

    // _________________OPEN_FILE____________________
    int32_t *res;           // output gpu
    FILE * handle;
    if (!(handle = fopen(file,"rb"))){
        printf("file not found!\n");
        return 0;
    }

    // Get file size
    fseek(handle, 0, SEEK_END);   // Move to the end of the file
    long fileSize = ftell(handle); // Get the current byte offset in the file
    fseek(handle, 0, SEEK_SET);   // Move back to the beginning of the file

    // printf("File size: %ld bytes\n", fileSize);
    int chunks_count = (fileSize + BUFSIZE -1) / BUFSIZE;

    // 
    // struct resultStructGJSON{
    //     std::vector<int> resultSizes;
    //     int totalResultSize;
    // };

    // printf("chunks_count: %d Chunks \n", chunks_count);

    hipHostMalloc(&buf, sizeof(uint8_t)*BUFSIZE);                          // input (each chunk)
    hipHostMalloc(&res_buf, sizeof(uint32_t)*BUFSIZE*chunks_count*ROW2);   // output(all chunks together)


    resultStruct->chunkCount = chunks_count;
    resultStruct->structural = res_buf;
    resultStruct->pair_pos = res_buf + BUFSIZE*chunks_count;

    // resultStruct->inputJSON = handle;

    // _________________READ_FILE____________________
    // Start  definition:
    ssize_t  read;
    uint8_t  *line = NULL;
    size_t   len = 0;
    uint32_t total = 0;
    uint32_t lines = 0;
    uint32_t lineLengths[1<<20]; //the maximum size of the array // we can convert it to 1 instead of array or remove it

    // //read start of file
    int i = 0;
    // int current_chunk_num = 0;
    int total_result_size = 0;          // latest index structural
    int latest_index_realJSON = 0;      // latest index realJSON
    while((read = getline((char **)&line, &len, handle)) != -1){        
        int readLimit = total + read;
        if(readLimit > BUFSIZE){
            // cout << current_chunk_num << endl;
            inputStartStruct inputStart;
            inputStart.block = buf;
            inputStart.size  = lineLengths[i-1];
            inputStart.lastChunkIndex = latest_index_realJSON;
            inputStart.lastStructuralIndex = total_result_size;
            res = (int32_t*) start( (void*) &inputStart);

            // device to host time:
            // hipEvent_t startDtoH, stopDtoH;
            // hipEventCreate(&startDtoH);
            // hipEventCreate(&stopDtoH);
            // hipEventRecord(startDtoH, 0);
            
            hipMemcpy(res_buf + 1 + total_result_size,                            res,                          sizeof(int32_t)*(inputStart.result_size), hipMemcpyDeviceToHost); // first and last is for [ and ]
            hipMemcpy(res_buf + 1 + BUFSIZE * chunks_count + total_result_size,   res + inputStart.result_size, sizeof(int32_t)*(inputStart.result_size), hipMemcpyDeviceToHost); // first and last is for [ and ]

            // cout << "sizeeee: " << inputStart.result_size << endl;
            total_result_size += inputStart.result_size;
            (resultStruct->resultSizesPrefix).push_back(total_result_size);
            (resultStruct->resultSizes).push_back(inputStart.result_size);


            // hipEventRecord(stopDtoH, 0);
            // hipEventSynchronize(stopDtoH);
            // float elapsedTime;
            // hipEventElapsedTime(&elapsedTime, startDtoH, stopDtoH);
            // time_EE.copy_end += elapsedTime;
            
            hipFree(res);
            res = res_buf;

            hipDeviceSynchronize();
            
            
            latest_index_realJSON += total;
            total = 0;
            // totalRecord += i;
            i = 0;

            memcpy(buf+total, line, sizeof(uint8_t)*read);
            total = read; //Reset

            // totalChar += read;

            lineLengths[i] = total;


        }else{
            memcpy(buf+total, line, sizeof(uint8_t)*read);
            total += read;
            // totalChar += read;
            //printf("size before star: %d \n",total);
            lineLengths[i] = total;
        }
        i++;

    }

    // remaining parts that aree very small
    if(total > 0){
        //print8(buf, total, ROW1);
        inputStartStruct inputStart;
        inputStart.block = buf;
        inputStart.size = lineLengths[i-1];
        inputStart.lastChunkIndex = latest_index_realJSON;
        inputStart.lastStructuralIndex = total_result_size;

        // printf("remaining injast\n");
        //printf("%s \n",buf);
        res = (int32_t*) start( (void*) &inputStart);
        //printf("remaining injast 2\n");


        // hipEvent_t startDtoH, stopDtoH;
        // hipEventCreate(&startDtoH);
        // hipEventCreate(&stopDtoH);
        // hipEventRecord(startDtoH, 0);
        

            

        hipMemcpy(res_buf + 1 + total_result_size,                            res,                          sizeof(int32_t)*(inputStart.result_size), hipMemcpyDeviceToHost);
        hipMemcpy(res_buf + 1 + total_result_size + BUFSIZE * chunks_count,   res + inputStart.result_size, sizeof(int32_t)*(inputStart.result_size), hipMemcpyDeviceToHost);



        // cout << "sizeeee: " << inputStart.result_size << endl;
        total_result_size += inputStart.result_size;
        (resultStruct->resultSizesPrefix).push_back(total_result_size);
        (resultStruct->resultSizes).push_back(inputStart.result_size);
        // cout << "total size = " << total_result_size << endl;
        // for (int i = 0; i < inputStart.result_size; i++){
        //     cout << i << "-->" << res_buf[i] << "\t";
        // }
        // cout << endl;
        // cout << endl;
        // for (int i = 0; i < inputStart.result_size; i++){
        //     cout << i << "-->" << (res_buf + BUFSIZE)[i] << "\t";
        // }
        // cout << endl;
        // cout << endl;
        // for (int i = 0; i < totalResultSize; i++){
        //     cout << i << "-2->" << parsedTree.pair_pos[i] << "\t";
        // }
        // cout << endl;
        // current_chunk_num++;

        // hipEventRecord(stopDtoH, 0);
        // hipEventSynchronize(stopDtoH);
        // float elapsedTime;
        // hipEventElapsedTime(&elapsedTime, startDtoH, stopDtoH);
        // time_EE.copy_end += elapsedTime;
            

        hipFree(res);
        res = res_buf;
        latest_index_realJSON += total;
        // print32(res,inputStart.result_size,ROW3);
        //puts(res);
        hipDeviceSynchronize();
        
    }

    total = 0;

    // hipHostFree(res_buf);
    hipHostFree(buf);
    fclose(handle);


    //t cout << "Total Query    : " << time_cal.query_t <<endl;
    //t cout << "Total Validation: " << time_cal.validation_t <<endl;
    //t cout << "Total Tokenizer : " << time_cal.tokenizer_t <<endl;
    //t cout << "Total Parser    : " << time_cal.parser_t <<endl;

    //t time_cal.validation_t = 0;
    //t time_cal.tokenizer_t = 0;
    //t time_cal.parse_tree = 0;



    // if(n == 6){
    //     cout << "Warmup HtoD Time:" << time_EE.copy_start << endl;
    //     cout << "Warmup Start Running: " << time_EE.EE_t <<endl;
    //     cout << "Warmup DtoH Time:" << time_EE.copy_end << endl;
    // }else{
    //     cout << "Attempt "<< (6-n) << "th for copying HtoD:" << time_EE.copy_start<<endl;
    //     cout << "Attempt "<< (6-n) << "th for running:" << time_EE.EE_t<<endl;
    //     cout << "Attempt "<< (6-n) << "th for copying DtoH:" << time_EE.copy_end<<endl;
    //     time_EE.EE_total += time_EE.EE_t;
    //     time_EE.copy_end_toal += time_EE.copy_end;
    //     time_EE.copy_start_total += time_EE.copy_start;
    // }
    // time_EE.EE_t = 0;
    // time_EE.copy_end = 0;
    // time_EE.copy_start = 0;

    resultStruct->totalResultSize = total_result_size + 2;
    resultStruct->fileSize = latest_index_realJSON + 2;
    return res;
}

// user side: LOL
int main(int argc, char **argv){
    int32_t* result;
    if (argv[1] != NULL){
        if( strcmp(argv[1], "-b") == 0 && argv[2] != NULL){
            std::cout << "Batch mode running..." << std::endl;
            int n = 6;
            float total_time = 0;

            resultStructGJSON parsed_tree; 
            parsed_tree.bufferSize = BUFSIZE;
            parsed_tree.chunkCount = 0;
            parsed_tree.totalResultSize = 0;
            parsed_tree.resultSizes;
            parsed_tree.resultSizesPrefix;
            parsed_tree.structural = NULL;
            parsed_tree.pair_pos = NULL;

            result = readFileLine(argv[2], 1 , &parsed_tree);

            int index0;
            high_resolution_clock::time_point start, stop;

            structural_iterator itr = structural_iterator(&parsed_tree,argv[2]);
            index0 = itr.gotoArrayIndex(2);
            index0 = itr.gotoKey("descriptions");
            itr.reset();

            start = high_resolution_clock::now();
            //WM
            index0 = itr.gotoArrayIndex(2);
            index0 = itr.gotoKey("descriptions");

            stop = high_resolution_clock::now();
            auto elapsed = duration_cast<nanoseconds>(stop - start);
            cout << "\nValue: " << itr.getValue() <<endl;
            cout << "Total Query time: " << elapsed.count() << " nanoseconds." << endl << endl;
            itr.freeJson();

            
            hipHostFree(parsed_tree.structural);

           
        }
        else std::cout << "Command should be like '-b[file path]'" << std::endl;
    }
    else{
        std::cout << "Please select (batch: -b): " << std::endl;
    } 
    hipDeviceReset();
    return 0;
}