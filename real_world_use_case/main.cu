#include "../cujson/cujsonlines.h"

int main(int argc, char **argv) {
    std::string filePath = "../dataset/merged_output.json";
    // Check command-line arguments
    if (argc >= 2) {
        filePath = argv[1];  // XML file path
        cout << "\033[1;36m[INFORM]\033[0m Using custom JSON file from command line: " << filePath << "\n";
    } else {
        std::cout << "\033[1;36m[INFORM]\033[0m Using default JSON file path.\n";
    }



    // Load File
    size_t maxChunkSizeMB = 512; // 256MB
    cuJSONLinesInput input = loadJSONLines_chunkSizeMegaBytes(filePath, maxChunkSizeMB);
    if (!input.data) {
        std::cout << "\033[1;31m[ERR]\033[0m File loading failed. Please check the file path.\n";
        return EXIT_FAILURE;
    }
    
    cout << "\033[1;32m[INFORM]\033[0m File loaded successfully. Size: " << input.size << " bytes\n";
    cout << "\033[1;32m[INFORM]\033[0m Number of chunks: " << input.chunkCount << "\n";
    cout << "\033[1;32m[INFORM]\033[0m Chunk sizes: ";
    for (size_t i = 0; i < input.chunksSize.size(); ++i) {
        std::cout << input.chunksSize[i] << " ";
    }
    std::cout << "\n";  


    // Parse JSON with cuJSON
    
    // report time with chrono in milliseconds:
    auto start = std::chrono::high_resolution_clock::now();
    cuJSONResult parsed_tree = parse_json_lines(input);
    if(parsed_tree.structural == nullptr) {
        std::cout << "\033[1;31m[ERR]\033[0m JSON parsing failed.\n";
        hipHostFree(input.data);
        return EXIT_FAILURE;
    }
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);





    // Process the parsed tree as needed
    // For example, you can print the parsed tree size or perform further operations
    std::cout << "\033[1;32m[RESULT]\033[0m Parsed tree size: " << parsed_tree.totalResultSize << " elements\n";


    // print values of parsed_tree.structural
    // std::cout << "\033[1;32m[RESULT]\033[0m Parsed tree structural values:\n";
    // for (int i = 0; i < parsed_tree.totalResultSize && i < 150; ++i) {
    //     std::cout << parsed_tree.structural[i] << " ";
    // }
    // std::cout << "\n";
    
        
    // // Or you can traverse it for the query purpose:

    // report time with chrono in milliseconds:

    cuJSONLinesIterator itr = cuJSONLinesIterator(&parsed_tree, filePath.c_str());

    auto start_itr = std::chrono::high_resolution_clock::now();
    // //TT1
    int index0;
    index0 = itr.gotoArrayIndex(0);
    int count = 0;
    for(int i = 0; i < 211259; i++) {
    // for(int i = 0; i < 5; i++) {
        bool condition = itr.checkKeyValue("type", "PushEvent");
        if(condition == true){
            index0 = itr.gotoKey("id");
            count++;
            // std::cout << "\033[1;32m[RESULT]\033[0m Query Value of $[i].id: " << itr.getValue() << "\n";
        }
        // index0 = itr.gotoKey("id");
        // index0 = itr.gotoKey("repo");
        // index0 = itr.gotoKey("name");
        // itr.reset();
        index0 = itr.gotoNextSibling(1);

    } 
    // int fileLines = 2111259;

    // cuJSONLinesIterator itr = cuJSONLinesIterator(&parsed_tree, filePath.c_str());
    // int index0 = itr.gotoArrayIndex(0);
    // int count = 0;
    // for(int i = 0; i < fileLines; i++) {
    //     bool condition = itr.checkKeyValue("type", "PushEvent");
    //     if(condition == true){
    //         index0 = itr.gotoKey("repo");
    //         index0 = itr.gotoKey("name");            
    //         count++;
            
    //     }
    //     index0 = itr.gotoNextSibling(1);
    // } 

    auto end_itr = std::chrono::high_resolution_clock::now();
    auto duration_itr = std::chrono::duration_cast<std::chrono::nanoseconds>(end_itr - start_itr);

    cout << "\033[1;32m[RESULT]\033[0m Total PushEvent count: " << count << "\n";

    // std::cout << "\033[1;32m[RESULT]\033[0m Query Value of $[0].repo.name: " << itr.getValue() << "\n";


    // report time of parse and query and total time:
    std::cout << "\033[1;32m[RESULT]\033[0m Parsing time: " << duration.count() / 1e6 << " ms\n";
    std::cout << "\033[1;32m[RESULT]\033[0m Query time: " << duration_itr.count() / 1e6 << " ms\n";
    std::cout << "\033[1;32m[RESULT]\033[0m Total time: " << (duration.count() + duration_itr.count()) / 1e6 << " ms\n";      

    itr.freeJson();


    hipDeviceReset();
    return 0;
}