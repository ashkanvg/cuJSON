#include "load_file.h"
#include <fstream>
#include <sstream>
#include <iostream>


cuJSONInput loadJSON(const std::string& filePath) { 

    // ______________________LOAD_FILE_____________________________
    std::ifstream file(filePath, std::ios::binary | std::ios::ate);                     // Open in binary mode, seek to end
    if (!file) {                                                                        // unable to open file
        std::cerr << "\033[1;31m Error: Unable to open file: \033[0m \n" << filePath << std::endl;
        return {nullptr, 0};                                                            // Return nullptr and size 0
    }
    
    size_t fileSize = file.tellg();                                                     // Get file size
    file.seekg(0, std::ios::beg);                                                       // Seek back to start


    // allocate pinned memory (Host Memory)
    uint8_t* h_buffer;                                                                  // the place that we store it    
    hipHostAlloc((void**)&h_buffer, fileSize * sizeof(uint8_t), hipHostMallocDefault); // allocate pinned memory

    if (!h_buffer) {                                                                    // Unable to allocate pinned memory!
        std::cerr << "\033[1;31m Error: Unable to allocate pinned memory! \033[0m \n" << std::endl;
        file.close();           
        return {nullptr, 0};                                                            // Return nullptr and size 0
    }


    // Read file content into buffer
    file.read(reinterpret_cast<char*>(h_buffer), fileSize);                             // copy from memory to host
    file.close();                                   


    return {h_buffer, fileSize}; // Return the buffer and its size
}


cuJSONLinesInput loadJSONLines(const std::string& filePath, size_t chunkCount = 1) { 
    cuJSONLinesInput input; 
    input.data = nullptr;
    input.size = 0;
    input.chunkCount = chunkCount;

    // ______________________LOAD_FILE_____________________________
    std::ifstream file(filePath, std::ios::binary | std::ios::ate);                     // Open in binary mode, seek to end
    if (!file) {                                                                        // unable to open file
        std::cerr << "\033[1;31m Error: Unable to open file: \033[0m \n" << filePath << std::endl;
        return input;                                                                   // Return nullptr and size 0
    }
    
    size_t fileSize = file.tellg();                                                     // Get file size
    file.seekg(0, std::ios::beg);                                                       // Seek back to start


    // allocate pinned memory (Host Memory)
    uint8_t* h_buffer;                                                                  // the place that we store it    
    hipHostAlloc((void**)&h_buffer, fileSize * sizeof(uint8_t), hipHostMallocDefault); // allocate pinned memory

    if (!h_buffer) {                                                                    // Unable to allocate pinned memory!
        std::cerr << "\033[1;31m Error: Unable to allocate pinned memory! \033[0m \n" << std::endl;
        file.close();           
        return input;                                                                   // Return nullptr and size 0
    }


    // Read file content into buffer
    file.read(reinterpret_cast<char*>(h_buffer), fileSize);                             // copy from memory to host
    file.close();   
    

    // Find line offsets
    std::vector<size_t> line_offsets;
    line_offsets.push_back(0);  // Start of first line
    for (size_t i = 0; i < fileSize; ++i) {
        if (h_buffer[i] == '\n') {
            line_offsets.push_back(i + 1);  // Start of next line
        }
    }

    // If last line doesn’t end with \n, manually add EOF as offset
    if (line_offsets.back() < fileSize) line_offsets.push_back(fileSize);

    size_t total_lines = line_offsets.size() - 1;
    size_t lines_per_chunk = total_lines / chunkCount;
    size_t extra_lines = total_lines % chunkCount;

    input.data = h_buffer;
    input.size = fileSize;
    input.chunks.reserve(chunkCount);
    input.chunksSize.reserve(chunkCount);

    size_t line_idx = 0;
    for (size_t i = 0; i < chunkCount; ++i) {
        size_t lines_in_this_chunk = lines_per_chunk + (i < extra_lines ? 1 : 0);

        size_t start_offset = line_offsets[line_idx];
        size_t end_offset = line_offsets[line_idx + lines_in_this_chunk];

        input.chunks.push_back(h_buffer + start_offset);
        input.chunksSize.push_back(end_offset - start_offset);

        line_idx += lines_in_this_chunk;
    }

    
    return input; // Return the buffer and its size
}