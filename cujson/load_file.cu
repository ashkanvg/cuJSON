#include "load_file.h"
#include <fstream>
#include <sstream>
#include <iostream>


cuJSONInput loadJSON(const std::string& filePath) { 
    // ______________________LOAD_FILE_____________________________
    std::ifstream file(filePath, std::ios::binary | std::ios::ate);                     // Open in binary mode, seek to end
    if (!file) {                                                                        // unable to open file
        std::cerr << "\033[1;31m Error: Unable to open file: \033[0m \n" << filePath << std::endl;
        return {nullptr, 0};                                                            // Return nullptr and size 0
    }
    
    size_t fileSize = file.tellg();                                                     // Get file size
    file.seekg(0, std::ios::beg);                                                       // Seek back to start


    // allocate pinned memory (Host Memory)
    uint8_t* h_buffer;                                                                  // the place that we store it    
    hipHostAlloc((void**)&h_buffer, fileSize * sizeof(uint8_t), hipHostMallocDefault); // allocate pinned memory

    if (!h_buffer) {                                                                    // Unable to allocate pinned memory!
        std::cerr << "\033[1;31m Error: Unable to allocate pinned memory! \033[0m \n" << std::endl;
        file.close();           
        return {nullptr, 0};                                                            // Return nullptr and size 0
    }


    // Read file content into buffer
    file.read(reinterpret_cast<char*>(h_buffer), fileSize);                             // copy from memory to host
    file.close();                                   


    return {h_buffer, fileSize}; // Return the buffer and its size
}

cuJSONLinesInput loadJSONLines_chunkCount(const std::string& filePath, size_t chunkCount = 1) { 
    cuJSONLinesInput input; 
    input.data = nullptr;
    input.size = 0;
    input.chunkCount = chunkCount;

    // ______________________LOAD_FILE_____________________________
    std::ifstream file(filePath, std::ios::binary | std::ios::ate);                     // Open in binary mode, seek to end
    if (!file) {                                                                        // unable to open file
        std::cerr << "\033[1;31m Error: Unable to open file: \033[0m \n" << filePath << std::endl;
        return input;                                                                   // Return nullptr and size 0
    }
    
    size_t fileSize = file.tellg();                                                     // Get file size
    file.seekg(0, std::ios::beg);                                                       // Seek back to start


    // allocate pinned memory (Host Memory)
    uint8_t* h_buffer;                                                                  // the place that we store it    
    hipHostAlloc((void**)&h_buffer, fileSize * sizeof(uint8_t), hipHostMallocDefault); // allocate pinned memory

    if (!h_buffer) {                                                                    // Unable to allocate pinned memory!
        std::cerr << "\033[1;31m Error: Unable to allocate pinned memory! \033[0m \n" << std::endl;
        file.close();           
        return input;                                                                   // Return nullptr and size 0
    }


    // Read file content into buffer
    file.read(reinterpret_cast<char*>(h_buffer), fileSize);                             // copy from memory to host
    file.close();   
    

    // Find line offsets
    std::vector<size_t> line_offsets;
    line_offsets.push_back(0);  // Start of first line
    for (size_t i = 0; i < fileSize; ++i) {
        if (h_buffer[i] == '\n') {
            line_offsets.push_back(i + 1);  // Start of next line
        }
    }

    // If last line doesn’t end with \n, manually add EOF as offset
    if (line_offsets.back() < fileSize) line_offsets.push_back(fileSize);

    size_t total_lines = line_offsets.size() - 1;
    size_t lines_per_chunk = total_lines / chunkCount;
    size_t extra_lines = total_lines % chunkCount;

    input.data = h_buffer;
    input.size = fileSize;
    input.chunks.reserve(chunkCount);
    input.chunksSize.reserve(chunkCount);

    size_t line_idx = 0;
    for (size_t i = 0; i < chunkCount; ++i) {
        size_t lines_in_this_chunk = lines_per_chunk + (i < extra_lines ? 1 : 0);

        size_t start_offset = line_offsets[line_idx];
        size_t end_offset = line_offsets[line_idx + lines_in_this_chunk];

        input.chunks.push_back(h_buffer + start_offset);
        input.chunksSize.push_back(end_offset - start_offset);

        line_idx += lines_in_this_chunk;
    }

    
    return input; // Return the buffer and its size
}

cuJSONLinesInput loadJSONLines_chunkSizeBytes(const std::string& filePath, size_t chunkSizeBytes) {
    cuJSONLinesInput input;
    input.data = nullptr;
    input.size = 0;

    // ----------------- LOAD FILE -----------------
    std::ifstream file(filePath, std::ios::binary | std::ios::ate);
    if (!file) {
        std::cerr << "\033[1;31m Error: Unable to open file: \033[0m \n" << filePath << std::endl;
        return input;
    }

    size_t fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    // Allocate pinned memory
    uint8_t* h_buffer;
    hipHostAlloc((void**)&h_buffer, fileSize * sizeof(uint8_t), hipHostMallocDefault);
    if (!h_buffer) {
        std::cerr << "\033[1;31m Error: Unable to allocate pinned memory! \033[0m \n" << std::endl;
        file.close();
        return input;
    }

    // Read file
    file.read(reinterpret_cast<char*>(h_buffer), fileSize);
    file.close();

    // ----------------- FIND LINE OFFSETS -----------------
    std::vector<size_t> line_offsets;
    line_offsets.push_back(0);  // First line

    for (size_t i = 0; i < fileSize; ++i) {
        if (h_buffer[i] == '\n') {
            line_offsets.push_back(i + 1);  // Start of next line
        }
    }
    if (line_offsets.back() < fileSize) {
        line_offsets.push_back(fileSize);
    }

    input.data = h_buffer;
    input.size = fileSize;

    size_t current_chunk_start = 0;
    size_t current_offset = 0;
    size_t chunk_count = 0;
    for (size_t i = 1; i < line_offsets.size(); ++i) {
        size_t line_start = line_offsets[i - 1];
        size_t line_end = line_offsets[i];
        size_t line_len = line_end - line_start;

        // If adding this line would exceed the chunk size, finalize current chunk
        if ((line_end - current_chunk_start) > chunkSizeBytes) {
            chunk_count++;
            input.chunks.push_back(h_buffer + current_chunk_start);
            input.chunksSize.push_back(current_offset - current_chunk_start);
            current_chunk_start = line_start;
        }

        current_offset = line_end;
    }

    // Add last chunk
    if (current_chunk_start < fileSize) {
        input.chunks.push_back(h_buffer + current_chunk_start);
        input.chunksSize.push_back(fileSize - current_chunk_start);
        chunk_count++;
    }

    input.chunkCount = chunk_count;
    return input;
}

cuJSONLinesInput loadJSONLines_chunkSizeMegaBytes(const std::string& filePath, size_t chunkSizeMegaBytes) {
    cuJSONLinesInput input;
    input.data = nullptr;
    input.size = 0;
    size_t chunkSizeBytes = chunkSizeMegaBytes * 1024 * 1024; // Convert MB to bytes

    // ----------------- LOAD FILE -----------------
    std::ifstream file(filePath, std::ios::binary | std::ios::ate);
    if (!file) {
        std::cerr << "\033[1;31m Error: Unable to open file: \033[0m \n" << filePath << std::endl;
        return input;
    }

    size_t fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    // Allocate pinned memory
    uint8_t* h_buffer;
    hipHostAlloc((void**)&h_buffer, fileSize * sizeof(uint8_t), hipHostMallocDefault);
    if (!h_buffer) {
        std::cerr << "\033[1;31m Error: Unable to allocate pinned memory! \033[0m \n" << std::endl;
        file.close();
        return input;
    }

    // Read file
    file.read(reinterpret_cast<char*>(h_buffer), fileSize);
    file.close();

    // ----------------- FIND LINE OFFSETS -----------------
    std::vector<size_t> line_offsets;
    line_offsets.push_back(0);  // First line

    for (size_t i = 0; i < fileSize; ++i) {
        if (h_buffer[i] == '\n') {
            line_offsets.push_back(i + 1);  // Start of next line
        }
    }
    if (line_offsets.back() < fileSize) {
        line_offsets.push_back(fileSize);
    }

    input.data = h_buffer;
    input.size = fileSize;

    size_t current_chunk_start = 0;
    size_t current_offset = 0;
    size_t chunk_count = 0;
    for (size_t i = 1; i < line_offsets.size(); ++i) {
        size_t line_start = line_offsets[i - 1];
        size_t line_end = line_offsets[i];
        size_t line_len = line_end - line_start;

        // If adding this line would exceed the chunk size, finalize current chunk
        if ((line_end - current_chunk_start) > chunkSizeBytes) {
            chunk_count++;
            input.chunks.push_back(h_buffer + current_chunk_start);
            input.chunksSize.push_back(current_offset - current_chunk_start);
            current_chunk_start = line_start;
        }

        current_offset = line_end;
    }

    // Add last chunk
    if (current_chunk_start < fileSize) {
        input.chunks.push_back(h_buffer + current_chunk_start);
        input.chunksSize.push_back(fileSize - current_chunk_start);
        chunk_count++;
    }

    input.chunkCount = chunk_count;
    return input;
}
