#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <thread>
#include <x86intrin.h>
#include <string.h>
#include <bitset>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/gather.h>
#include <thrust/extrema.h>
#include <thrust/partition.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <inttypes.h>
#include <thrust/host_vector.h>
#include <>
#include <vector>
#include <cstring>
#include <hipcub/hipcub.hpp>




// #include "./12-GJSON-Class.cuh"

#define         MAXLINELENGTH     1073741824   //4194304 8388608 33554432 67108864 134217728 201326592 268435456 536870912 805306368 1073741824// Max record size
                                              //4MB       8MB     32BM    64MB      128MB    192MB     256MB     512MB     768MB       1GB
#define         BUFSIZE           1073741824   //4194304 8388608 33554432 67108864 134217728 201326592 268435456 536870912 805306368 1073741824

#define BLOCKSIZE 256

#define OPENBRACKET 91
#define CLOSEBRACKET 93
#define OPENBRACE 123
#define CLOSEBRACE 125
#define I 73

#define ROW1 1
#define ROW2 2
#define ROW3 3
#define ROW4 4
#define ROW5 5

#ifndef DEBUG_MODE
    #define DEBUG_MODE 1 
    // Set to 5 for debugging (memory consumption),
    // Set to 4 for debugging (size),
    // Set to 3 for debugging (overall time report), 
    // Set to 2 for debugging (time report), 
    // Set to 1 for debugging (print), 0 for production
#endif

using namespace std;
using namespace std::chrono;

#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip> // For formatting

void printGpuMemoryUsage(const std::string& msg = "") {
    size_t free_mem = 0;
    size_t total_mem = 0;

    hipError_t err = hipMemGetInfo(&free_mem, &total_mem);
    if (err != hipSuccess) {
        std::cerr << "hipMemGetInfo failed: " << hipGetErrorString(err) << std::endl;
        return;
    }

    size_t used_mem = total_mem - free_mem;

    std::cout << std::fixed << std::setprecision(2);
    if (!msg.empty()) {
        std::cout << "[" << msg << "] \n";
    }
    std::cout << "GPU Memory Usage: "
              << "Used = " << used_mem / (1024.0 * 1024.0) << " MB, "
              << "Free = " << free_mem / (1024.0 * 1024.0) << " MB, "
              << "Total = " << total_mem / (1024.0 * 1024.0) << " MB" << std::endl;
}



// Struct to check if a given integer is equal to 1.
struct is_one {
    __host__ __device__ // Can be called from both host (CPU) and device (GPU) code.
    bool operator()(const int x) { 
        return (x == 1); // Returns true if x is 1.
    }
};

// Struct to check if a character is an opening brace or bracket.
struct is_opening {
    __host__ __device__ // Can be called from both host (CPU) and device (GPU) code.
    bool operator()(char x) {
        return (x == OPENBRACE) || (x == OPENBRACKET); // Returns true for '{' or '['.
    }
};

// Struct to check if a character is a closing brace or bracket.
struct is_closing {
    __host__ __device__ // Can be called from both host (CPU) and device (GPU) code.
    bool operator()(char x) {
        return (x == CLOSEBRACE) || (x == CLOSEBRACKET); // Returns true for '}' or ']'.
    }
};

// Struct to decrease an integer by 1.
struct decrease {
    __host__ __device__ // Can be called from both host (CPU) and device (GPU) code.
    int operator()(int x) {
        return x - 1; // Decreases the input integer by 1.
    }
};

// Struct to increase an integer by 1.
struct increase {
    __host__ __device__ // Can be called from both host (CPU) and device (GPU) code.
    int operator()(int x) {
        return x + 1; // Increases the input integer by 1.
    }
};





// Inline device function to compute the prefix XOR for a 32-bit integer.
// This function performs XOR-based prefix computations for efficiency.
__device__ __forceinline__
uint32_t prefix_xor(uint32_t x) {
    x ^= (x << 1);   // XOR with left-shifted version by 1 bit.
    x ^= (x << 2);   // XOR with left-shifted version by 2 bits.
    x ^= (x << 4);   // XOR with left-shifted version by 4 bits.
    x ^= (x << 8);   // XOR with left-shifted version by 8 bits.
    x ^= (x << 16);  // XOR with left-shifted version by 16 bits.
    return x;        // Returns the resulting XOR value.
}

// Inline device function to compute the prefix XOR for a 64-bit integer.
// This function performs XOR-based prefix computations for efficiency.
__device__ __forceinline__
uint64_t prefix_xor64(uint64_t x) {
    x ^= (x << 1);   // XOR with left-shifted version by 1 bit.
    x ^= (x << 2);   // XOR with left-shifted version by 2 bits.
    x ^= (x << 4);   // XOR with left-shifted version by 4 bits.
    x ^= (x << 8);   // XOR with left-shifted version by 8 bits.
    x ^= (x << 16);  // XOR with left-shifted version by 16 bits.
    x ^= (x << 32);  // XOR with left-shifted version by 32 bits.
    return x;        // Returns the resulting XOR value.
}




// _______________________Debug__Functions_______________________

// _______________________Device_Functions_______________________
// Converts a single byte (8 bits) into its binary string representation.
// Returns a pointer to a shared memory string containing the binary representation.
// __device__
// const char* byteToBinary(uint8_t byte) {
//     __shared__ char binary[9]; // Shared memory buffer for binary string (ensure no race conditions).
//     binary[8] = '\0'; // Null terminator for the binary string.

//     for (int i = 7; i >= 0; --i) {
//         binary[i] = (byte & 0x01) ? '1' : '0'; // Extract the least significant bit.
//         byte >>= 1; // Shift right to process the next bit.
//     }

//     return binary;
// }

// // Converts a 32-bit unsigned integer into its binary string representation.
// // The output buffer `out` must have at least 33 characters (32 bits + null terminator).
// __device__
// void u32ToBinary(uint32_t num, char* out) {
//     out[32] = '\0'; // Null terminator for the binary string.
//     for (int i = 31; i >= 0; --i) {
//         out[i] = (num & 0x01) ? '1' : '0'; // Extract the least significant bit.
//         num >>= 1; // Shift right to process the next bit.
//     }
// }


// _______________________Host_Functions_for_Debugging_GPU_Data_______________________

// Print the first 100 characters of the XML file.
void printFirst100Chars(const uint8_t* data, size_t length) {
    if (!data || length == 0) {
        std::cerr << "\033[1;34m Warning: No data to print! \033[0m" << std::endl;
        return;
    }

    std::cout << "First 100 characters of XML file:\n";
    for (size_t i = 0; i < 100 && i < length; i++) {
        std::cout << static_cast<char>(data[i]);
    }
    std::cout << std::endl;
}

// Prints a 2D array of 32-bit unsigned integers stored on the GPU.
// Converts each value to its binary representation and outputs row by row.
void print_d32(uint32_t* d_data, int total_padded_32, int rows) {
    uint32_t* h_data = (uint32_t*)malloc(total_padded_32 * rows * sizeof(uint32_t)); // Host buffer.
    if (!h_data) {
        std::cerr << "Failed to allocate host memory!" << std::endl;
        return;
    }

    hipMemcpy(h_data, d_data, total_padded_32 * rows * sizeof(uint32_t), hipMemcpyDeviceToHost); // Copy from device to host.

    for (int i = 0; i < total_padded_32 * rows; ++i) {
        uint32_t value = h_data[i];
        for (int j = 0; j < 32; ++j) { // Print each bit of the value.
            std::cout << ((value >> j) & 1);
        }
        std::cout << std::endl;
    }

    free(h_data); // Free host memory.
}

// Prints selected portions of a 2D array of 32-bit unsigned integers stored on the GPU.
int print_d(uint32_t* input_GPU, int length, int rows) {
    uint32_t* input = (uint32_t*)malloc(sizeof(uint32_t) * length * rows); // Host buffer.
    hipMemcpyAsync(input, input_GPU, sizeof(uint32_t) * length * rows, hipMemcpyDeviceToHost); // Async copy to host.

    for (long i = 0; i < rows; i++) {
        for (long j = 401; j < 470 && j < length; j++) { // Print a specific range of columns.
            std::bitset<32> y(*(input + j + (i * length))); // Convert to binary using std::bitset.
            if (j == 129) printf("----129----");
            std::cout << y << ' ';
        }
        std::cout << "\n";
    }

    free(input); // Free host memory.
    return 1;
}

// Prints a 2D array of 8-bit unsigned integers stored on the host.
int print8(uint8_t* input, int length, int rows) {
    for (long i = 0; i < rows; i++) {
        for (long j = 0; j < length && j < 200; j++) { // Print up to 200 values per row.
            std::cout << *(input + j + (i * length)) << ' ';
        }
        std::cout << std::endl;
    }
    return 1;
}

// Prints a 2D array of 32-bit integers stored on the host.
int print32(int32_t* input, int length, int rows) {
    for (long i = 0; i < rows; i++) {
        for (long j = 0; j < length && j < 200; j++) { // Print up to 200 values per row.
            std::cout << *(input + j + (i * length)) << ' ';
        }
        std::cout << std::endl;
    }
    return 1;
}

// Template function to print a 2D array of 8-bit integers from the GPU.
// The array is transferred to the host before printing.
template<typename T>
int print8_d(uint8_t* input_GPU, int length, int rows) {
    uint8_t* input = (uint8_t*)malloc(sizeof(uint8_t) * length); // Host buffer.
    hipMemcpyAsync(input, input_GPU, sizeof(uint8_t) * length, hipMemcpyDeviceToHost); // Async copy to host.

    for (long i = 0; i < rows; i++) {
        for (long j = 0; j < 300 && j < length; j++) { // Print up to 300 values per row.
            std::cout << (T)*(input + j + (i * length)) << ' ';
        }
        std::cout << std::endl;
    }

    free(input); // Free host memory.
    return 1;
}


void printByteByByte(int32_t* data, int length) {
    for (int i = 0; i < length; ++i) {
        unsigned char* bytePointer = (unsigned char*)&data[i];
        for (int j = 0; j < sizeof(int32_t); ++j) {
            printf("%02x ", bytePointer[j]);
        }
        printf("\n");
    }
}

// Prints a 2D array of 32-bit unsigned integers stored on the GPU.
void print_token_array_as_bytes(const char* label, uint32_t* d_array, size_t length_uint32) {
    size_t length_bytes = length_uint32 * sizeof(uint32_t);
    uint8_t* h_bytes = new uint8_t[length_bytes];

    hipMemcpy(h_bytes, d_array, length_bytes, hipMemcpyDeviceToHost);

    std::cout << "=== " << label << " (bytes) ===\n";
    for (size_t i = 0; i < length_bytes && i < 50; i++) {
        printf("%3d ", h_bytes[i]);  // Print as decimal byte
        if ((i + 1) % 8 == 0) std::cout << "\n";
    }
    std::cout << "\n";

    delete[] h_bytes;
}

// Prints the first 100 token indices from a device array of uint32_t.
void print_uint32_indices(const char* label, const uint32_t* d_finalTokens, size_t tokens_count) {
    std::cout << "=== " << label << " (token indices) ===\n";

    // Copy to host
    uint32_t* h_tokens = new uint32_t[tokens_count];
    hipMemcpy(h_tokens, d_finalTokens, tokens_count * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Print 8 per line
    for (size_t i = 0; i < tokens_count && i < 50; ++i) {
        std::cout << h_tokens[i] << " ";
        if ((i + 1) % 8 == 0) std::cout << "\n";
    }
    std::cout << "\n";

    delete[] h_tokens;
}

// Prints the first 100 token indices and their corresponding character values from the device arrays.
void print_token_info(const char* label, const uint32_t* d_indices, const uint8_t* d_values, size_t count) {
    std::cout << "=== " << label << " (token indices : char) ===\n";

    std::vector<uint32_t> h_indices(count);
    std::vector<uint8_t> h_values(count);

    hipMemcpy(h_indices.data(), d_indices, count * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_values.data(), d_values, count * sizeof(uint8_t), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < count && i < 50; ++i) {
        std::cout << h_indices[i] << " : '" << static_cast<char>(h_values[i]) << "'\n";
    }
}

// Prints the byte map of a device array (uint8_t) for debugging purposes.
void print_byte_map(const char* label, const uint8_t* d_array, size_t length) {
    #if defined(DEBUG_MODE) && DEBUG_MODE == 1
        std::cout << "=== " << label << " (byte-by-byte) ===\n";
    
        // Allocate host memory and copy from device
        uint8_t* h_array = new uint8_t[length];
        hipMemcpy(h_array, d_array, length * sizeof(uint8_t), hipMemcpyDeviceToHost);
    
        // Print 8 values per line
        for (size_t i = 0; i < length && i < 50; ++i) {
            printf("%3d ", h_array[i]);
            if ((i + 1) % 8 == 0) std::cout << "\n";
        }
        if (length % 8 != 0) std::cout << "\n";
    
        delete[] h_array;
    #endif
}

// Prints the byte signs (1 for opening, -1 for closing) stored on the GPU.
void print_byte_signs(const char* label, const int8_t* d_signs, size_t count) {
    std::cout << "=== " << label << " (byte-by-byte) ===\n";
    
    int8_t* h = new int8_t[count];
    hipMemcpy(h, d_signs, count * sizeof(int8_t), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < count && i < 50; ++i) {
        printf("%3d ", (int)h[i]);
        if ((i + 1) % 8 == 0) std::cout << "\n";
    }

    if (count % 8 != 0) std::cout << "\n";

    delete[] h;
}

// Prints the depth array stored on the GPU for debugging purposes. 
void print_uint32_array(const char* label, const uint32_t* d_depth, size_t count) {
    std::cout << "=== " << label << " (word-by-word) ===\n";
    
    std::vector<uint32_t> h_depth(count);
    hipMemcpy(h_depth.data(), d_depth, count * sizeof(uint32_t), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < count && i < 650; ++i) {
        std::cout << (int) h_depth[i] << "\t"; // Print each depth value
        if ((i + 1) % 8 == 0) std::cout << "\n";
    }
    if (count % 8 != 0) std::cout << "\n";
}

void print_query_output(int8_t* d_output, int num_steps, int tokens_count, int K) {
    int total_size = num_steps * tokens_count;
    std::vector<int8_t> h_output(total_size);

    // Copy from device to host
    hipMemcpy(h_output.data(), d_output, total_size * sizeof(int8_t), hipMemcpyDeviceToHost);

    // Print first K rows per step
    std::cout << "=== Output Matrix (d=1, d=2, d=3) ===" << std::endl;
    // std::cout << "Output Matrix (First " << K << " rows per step):\n";
    for (int step = 0; step < num_steps; ++step) {
        std::cout << "Step" << step << ": "<<endl;
        for (int i = 0; i < std::min(K, tokens_count) && i < 650; ++i) {
            int8_t val = h_output[step * tokens_count + i];
            std::cout << (int)val << "\t";  // cast to int for readable output
        }
        std::cout << std::endl;
    }
}

void print_int_device_vector(const thrust::device_vector<int>& d_vec, const std::string& name, int K) {
    std::vector<int> h_vec(d_vec.size());
    hipMemcpy(h_vec.data(), thrust::raw_pointer_cast(d_vec.data()), h_vec.size() * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "=== " << name << " (First " << K << " elements) ===" << std::endl;
    for (int i = 0; i < std::min(K, (int)h_vec.size()); ++i) {
        std::cout << h_vec[i] << "\t";
    }
    std::cout << std::endl;
}


void print_parsed_query(const vector<vector<string>>& parsed) {
    const vector<string> headers = {
        "Tag Name", "Attr Name", "Attr Cond Op", "Tag Cond Op",
        "Tag Cond Val", "Attr Cond Val", "Query Depth", "Attr Cond Name", "Tag Cond Name", 
        "Index Name", "Index Cond Op", "Index Cond Val"
    };

    size_t num_cols = parsed[0].size();
    cout << "Parsed XPath:\n";
    for (size_t r = 0; r < parsed.size(); ++r) {
        cout << headers[r] << ":\t";
        for (size_t c = 0; c < num_cols; ++c) {
            cout << (parsed[r][c].empty() ? "∅" : parsed[r][c]) << "\t";
        }
        cout << endl;
    }
}



template <typename T>
void print_device_array(const T* d_array, size_t count, const std::string& label) {
    std::vector<T> h_array(count);
    hipMemcpy(h_array.data(), d_array, count * sizeof(T), hipMemcpyDeviceToHost);
    std::cout << label << ": " << endl;
    for (size_t i = 0; i < count && i < 650; ++i) {
        std::cout << static_cast<int>(h_array[i]) << "\t";
    }
    std::cout << std::endl;
}

template <typename InputIterator>
void print_thrust_iterator(InputIterator begin, size_t count, const std::string& label) {
    size_t to_copy = std::min(count, static_cast<size_t>(650));
    std::vector<typename thrust::iterator_traits<InputIterator>::value_type> host_vec(to_copy);
    thrust::copy(begin, begin + to_copy, host_vec.begin());

    std::cout << label << ": " << endl;
    for (size_t i = 0; i < to_copy; ++i) {
        std::cout << static_cast<int>(host_vec[i]) << "\t";
    }
    std::cout << std::endl;
}


// ______________________check_CUDA_______________________
// Function to check the status of a CUDA API call and handle errors if any.
// If the CUDA call fails, the function prints the error message and terminates the program.
void checkCuda(hipError_t result) {
    if (result != hipSuccess) { // Check if the CUDA call did not succeed.
        // Print the error message associated with the CUDA error.
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        // Exit the program with a non-zero status to indicate an error.
        exit(1);
    }
}

// CUB functions:

// count_ones_cub: Counts the number of ones in a device array using CUB.
// This function uses the CUB library to perform a parallel reduction on the device array.
// It first queries the temporary storage needed for the reduction, then allocates that storage,
// performs the reduction, and finally copies the result back to the host.
// The function returns the count of ones found in the device array.
// The input array is expected to be a device pointer to an array of uint8_t values.
// The length parameter specifies the number of elements in the array.
// The function returns the count of ones found in the device array.
uint32_t count_ones_cub(uint8_t* d_flags, size_t length){
    // 1. temporary‑storage query
    void*  d_temp  = nullptr;
    size_t temp_sz = 0;
    uint32_t* d_result;                      // device scalar result
    hipMalloc(&d_result, sizeof(uint32_t));

    hipcub::DeviceReduce::Sum(
        d_temp,     temp_sz,                 // temp buffer ptr & size
        d_flags,    d_result,                // in, out
        length);                             // # elements

    // 2. allocate temp storage and run the real reduction
    hipMalloc(&d_temp, temp_sz);

    hipcub::DeviceReduce::Sum(
        d_temp, temp_sz,
        d_flags, d_result,
        length);

    // 3. copy result back to host
    uint32_t h_count = 0;
    hipMemcpy(&h_count, d_result, sizeof(uint32_t), hipMemcpyDeviceToHost);

    // 4. clean‑up
    hipFree(d_temp);
    hipFree(d_result);

    return h_count;
}

uint32_t reduce_cub_int(int8_t* d_flags, size_t length){
    // 1. temporary‑storage query
    void*  d_temp  = nullptr;
    size_t temp_sz = 0;
    uint32_t* d_result;                      // device scalar result
    hipMalloc(&d_result, sizeof(uint32_t));

    hipcub::DeviceReduce::Sum(
        d_temp,     temp_sz,                 // temp buffer ptr & size
        d_flags,    d_result,                // in, out
        length);                             // # elements

    // 2. allocate temp storage and run the real reduction
    hipMalloc(&d_temp, temp_sz);

    hipcub::DeviceReduce::Sum(
        d_temp, temp_sz,
        d_flags, d_result,
        length);

    // 3. copy result back to host
    uint32_t h_count = 0;
    hipMemcpy(&h_count, d_result, sizeof(uint32_t), hipMemcpyDeviceToHost);

    // 4. clean‑up
    hipFree(d_temp);
    hipFree(d_result);

    return h_count;
}



// inclusive_scan_inplace_cub: Performs an inclusive scan on a device array using CUB.
// This function uses the CUB library to perform an inclusive scan (prefix sum) on the input array.
// The input array is expected to be a device pointer to an array of int8_t values.
// The length parameter specifies the number of elements in the array.
// The function modifies the input array in place, storing the result back in the same array.
// The function does not return any value.
// It allocates temporary storage for the scan operation, performs the scan, and then frees the temporary storage.
// The input array is expected to be a device pointer to an array of int8_t values.
// The length parameter specifies the number of elements in the array.
// The function modifies the input array in place, storing the result back in the same array.
// The function does not return any value.
// It allocates temporary storage for the scan operation, performs the scan, and then frees the temporary storage.
// The function is designed to be efficient and uses CUB's parallel algorithms for the scan operation.
void inclusive_scan_inplace_cub(int8_t* d_data, size_t length) {
    // Allocate temp buffer
    void* d_temp = nullptr;
    size_t temp_bytes = 0;

    // First call: query temp storage
    hipcub::DeviceScan::InclusiveSum(
        d_temp, temp_bytes,
        d_data, d_data,  // in-place
        length
    );

    // Allocate temp buffer
    hipMalloc(&d_temp, temp_bytes);

    // Second call: actual inclusive scan
    hipcub::DeviceScan::InclusiveSum(
        d_temp, temp_bytes,
        d_data, d_data,
        length
    );

    // Free temp storage
    hipFree(d_temp);
}



// A small helper to count “1”s in a virtual flag stream using CUB
template<class FlagIter>
uint32_t count_virtual_flags_cub(FlagIter flags, size_t length ) {
    // device storage for the result
    uint32_t* d_result = nullptr;
    hipMalloc(&d_result, sizeof(uint32_t));

    // 1a) query temp storage size
    void*   d_temp   = nullptr;
    size_t  temp_sz  = 0;
    hipcub::DeviceReduce::Sum(
        d_temp, temp_sz,
        flags,    // could be a transform_iterator
        d_result, // device scalar
        length
    );

    // 1b) allocate temp storage & run real reduction
    hipMalloc(&d_temp, temp_sz);
    hipcub::DeviceReduce::Sum(
        d_temp, temp_sz,
        flags,
        d_result,
        length
    );

    // 1c) copy back
    uint32_t h_result = 0;
    hipMemcpy(&h_result, d_result, sizeof(uint32_t), hipMemcpyDeviceToHost);

    // clean up
    hipFree(d_temp);
    hipFree(d_result);

    return h_result;
}


// Copies elements from d_token_indices to d_selected_token_indices where d_output_flag == 1
// This function uses CUB's DeviceSelect::Flagged to perform the scatter operation.
// The input array d_token_indices is expected to be a device pointer to an array of uint32_t values.
// The d_output_flag array is a device pointer to an array of uint8_t values, where 1 indicates the element should be copied.
// The d_selected_token_indices array is a device pointer to an array of uint32_t values, where the selected elements will be copied.
// The tokens_count parameter specifies the number of elements in the input array.
// The function does not return any value.
// It allocates temporary storage for the scatter operation, performs the scatter, and then frees the temporary storage.
// The function is designed to be efficient and uses CUB's parallel algorithms for the scatter operation.

void scatter_cub(
    const uint32_t* d_token_indices,        // input data
    const uint8_t* d_output_flag,           // 0/1 flag for selection
    uint32_t* d_selected_token_indices,     // output buffer (preallocated)
    size_t tokens_count                     // number of input items
) {
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    uint32_t* d_num_selected_out;

    // Allocate temporary output count
    hipMalloc(&d_num_selected_out, sizeof(uint32_t));

    // Step 1: Query temporary storage size
    hipcub::DeviceSelect::Flagged(
        d_temp_storage, temp_storage_bytes,
        d_token_indices,            // input values
        d_output_flag,              // stencil
        d_selected_token_indices,   // output
        d_num_selected_out,         // number selected
        tokens_count
    );

    // Step 2: Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Step 3: Run the actual selection
    hipcub::DeviceSelect::Flagged(
        d_temp_storage, temp_storage_bytes,
        d_token_indices,
        d_output_flag,
        d_selected_token_indices,
        d_num_selected_out,
        tokens_count
    );

    // Optional: get count back (if you want to use it later)
    // uint32_t h_selected_count = 0;
    // hipMemcpy(&h_selected_count, d_num_selected_out, sizeof(uint32_t), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_temp_storage);
    hipFree(d_num_selected_out);
}


template<class FlagIter>
void scatter_virtual_flag_cub(
    const uint32_t*        d_token_indices,         // input indices
    size_t                 tokens_count,           // number of tokens
    FlagIter               flags,                  // transform_iterator over [0..tokens_count)
    uint32_t*              d_selected_tokens,      // OUT: compacted indices
    uint32_t*              d_selected_count       // OUT: device scalar count
) {
    // 2a) query temp storage
    void*  d_temp  = nullptr;
    size_t temp_sz = 0;
    auto  index_begin = thrust::make_counting_iterator<uint32_t>(0);

    hipcub::DeviceSelect::Flagged(
       d_temp, temp_sz,
       index_begin,     // the “input items” (we really only care about the index)
       flags,           // the lazy flag stream
       d_selected_tokens,
       d_selected_count,
       tokens_count
    );

    // 2b) allocate & run
    hipMalloc(&d_temp, temp_sz);
    hipcub::DeviceSelect::Flagged(
       d_temp, temp_sz,
       index_begin,
       flags,
       d_selected_tokens,
       d_selected_count,
       tokens_count
    );
    hipFree(d_temp);
}



