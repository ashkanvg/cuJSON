#include "cujson/cujson.h"


int main(int argc, char **argv) {

    std::string filePath = "./dataset/twitter_sample_large_record.json";
    // Check command-line arguments
    if (argc >= 2) {
        filePath = argv[1];  // XML file path
        cout << "\033[1;36m[INFORM]\033[0m Using custom JSON file from command line: " << filePath << "\n";
    } else {
        std::cout << "\033[1;36m[INFORM]\033[0m Using default JSON file path.\n";
    }


    // Load File
    cuJSONInput input = loadJSON(filePath);
    if (!input.data) {
        std::cout << "\033[1;31m[ERR]\033[0m File loading failed. Please check the file path.\n";
        return EXIT_FAILURE;
    }
    

    // Parse JSON with cuJSON
    auto start_time = std::chrono::high_resolution_clock::now();
    cuJSONResult parsed_tree = parse_standard_json(input);
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end_time - start_time;
    std::cout << "parse_standard_json took " << elapsed.count() << " ms" << std::endl;


    if(parsed_tree.structural == nullptr) {
        std::cout << "\033[1;31m[ERR]\033[0m JSON parsing failed.\n";
        hipHostFree(input.data);
        return EXIT_FAILURE;
    }


    // Process the parsed tree as needed
    // For example, you can print the parsed tree size or perform further operations
    std::cout << "\033[1;32m[RESULT]\033[0m Parsed tree size: " << parsed_tree.totalResultSize << " elements\n";


        
    // Or you can traverse it for the query purpose:
    cuJSONIterator itr = cuJSONIterator(&parsed_tree, filePath.c_str());

    //TT1
    // int index0;
    // index0 = itr.gotoArrayIndex(0);
    // index0 = itr.gotoArrayIndex(0);
    // index0 = itr.gotoKey("user");
    // index0 = itr.gotoKey("lang");
    // itr.reset();
    // index0 = itr.gotoArrayIndex(0);
    // index0 = itr.gotoArrayIndex(0);
    // index0 = itr.gotoKey("lang");

    // std::cout << "\033[1;32m[RESULT]\033[0m Query Value of $[0].lang: " << itr.getValue() << "\n";
    itr.freeJson();


    hipDeviceReset();
    return 0;
}