#include "cujson/cujsonlines.h"

int main(int argc, char **argv) {
    std::string filePath = "./dataset/twitter_sample_small_records.json";
    // Check command-line arguments
    if (argc >= 2) {
        filePath = argv[1];  // XML file path
        cout << "\033[1;36m[INFORM]\033[0m Using custom JSON file from command line: " << filePath << "\n";
    } else {
        std::cout << "\033[1;36m[INFORM]\033[0m Using default JSON file path.\n";
    }



    // Load File
    cuJSONLinesInput input = loadJSONLines_chunkCount(filePath, 4);
    if (!input.data) {
        std::cout << "\033[1;31m[ERR]\033[0m File loading failed. Please check the file path.\n";
        return EXIT_FAILURE;
    }
    
    cout << "\033[1;32m[INFORM]\033[0m File loaded successfully. Size: " << input.size << " bytes\n";
    cout << "\033[1;32m[INFORM]\033[0m Number of chunks: " << input.chunkCount << "\n";
    cout << "\033[1;32m[INFORM]\033[0m Chunk sizes: ";
    for (size_t i = 0; i < input.chunksSize.size(); ++i) {
        std::cout << input.chunksSize[i] << " ";
    }
    std::cout << "\n";  


    // Parse JSON with cuJSON
    cuJSONResult parsed_tree = parse_json_lines(input);
    if(parsed_tree.structural == nullptr) {
        std::cout << "\033[1;31m[ERR]\033[0m JSON parsing failed.\n";
        hipHostFree(input.data);
        return EXIT_FAILURE;
    }



    // Process the parsed tree as needed
    // For example, you can print the parsed tree size or perform further operations
    std::cout << "\033[1;32m[RESULT]\033[0m Parsed tree size: " << parsed_tree.totalResultSize << " elements\n";


    // print values of parsed_tree.structural
    // std::cout << "\033[1;32m[RESULT]\033[0m Parsed tree structural values:\n";
    // for (int i = 0; i < parsed_tree.totalResultSize && i < 150; ++i) {
    //     std::cout << parsed_tree.structural[i] << " ";
    // }
    // std::cout << "\n";
    
        
    // // Or you can traverse it for the query purpose:
    cuJSONLinesIterator itr = cuJSONLinesIterator(&parsed_tree, filePath.c_str());

    // //TT1
    int index0;
    index0 = itr.gotoArrayIndex(0);
    index0 = itr.gotoKey("lang");
    std::cout << "\033[1;32m[RESULT]\033[0m Query Value of $[0].user.lang: " << itr.getValue() << "\n";

    itr.freeJson();


    hipDeviceReset();
    return 0;
}